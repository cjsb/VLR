#include "hip/hip_runtime.h"
#include "kernel_common.cuh"
#include "random_distributions.cuh"

namespace VLR {
    // per Material
    rtDeclareVariable(progSigBSDFMatches, pv_progBSDFmatches, , );
    rtDeclareVariable(progSigSampleBSDFInternal, pv_progSampleBSDFInternal, , );
    rtDeclareVariable(progSigEvaluateBSDFInternal, pv_progEvaluateBSDFInternal, , );
    rtDeclareVariable(progSigEvaluateBSDF_PDFInternal, pv_progEvaluateBSDF_PDFInternal, , );
    rtDeclareVariable(progSigEvaluateEmittance, pv_progEvaluateEmittance, , );
    rtDeclareVariable(progSigEvaluateEDFInternal, pv_progEvaluateEDFInternal, , );

    // Should texCoord be SurfacePoint instead of TexCoord2D? or dedicated type for identifying a point?
    RT_FUNCTION RGBSpectrum sampleBSDF(const TexCoord2D &texCoord, const BSDFQuery &query, const BSDFSample &sample, BSDFQueryResult* result) {
        if (!pv_progBSDFmatches(query.dirTypeFilter)) {
            result->dirPDF = 0.0f;
            result->sampledType = DirectionType();
            return RGBSpectrum::Zero();
        }
        RGBSpectrum fs_sn = pv_progSampleBSDFInternal(texCoord, query, sample.uComponent, sample.uDir, result);
        float snCorrection = std::fabs(result->dirLocal.z / dot(result->dirLocal, query.geometricNormalLocal));
        return fs_sn * snCorrection;
    }

    RT_FUNCTION RGBSpectrum evaluateBSDF(const TexCoord2D &texCoord, const BSDFQuery &query, const Vector3D &dirLocal) {
        RGBSpectrum fs_sn = pv_progEvaluateBSDFInternal(texCoord, query, dirLocal);
        float snCorrection = std::fabs(dirLocal.z / dot(dirLocal, query.geometricNormalLocal));
        return fs_sn * snCorrection;
    }

    RT_FUNCTION float evaluateBSDF_PDF(const TexCoord2D &texCoord, const BSDFQuery &query, const Vector3D &dirLocal) {
        if (!pv_progBSDFmatches(query.dirTypeFilter)) {
            return 0;
        }
        float ret = pv_progEvaluateBSDF_PDFInternal(texCoord, query, dirLocal);
        return ret;
    }

    RT_FUNCTION RGBSpectrum evaluateEmittance(const TexCoord2D &texCoord) {
        RGBSpectrum Le0 = pv_progEvaluateEmittance(texCoord);
        return Le0;
    }

    RT_FUNCTION RGBSpectrum evaluateEDF(const TexCoord2D &texCoord, const EDFQuery &query, const Vector3D &dirLocal) {
        RGBSpectrum Le1 = pv_progEvaluateEDFInternal(texCoord, query, dirLocal);
        return Le1;
    }



    rtDeclareVariable(rtObject, pv_topGroup, , );
    rtDeclareVariable(DiscreteDistribution1D, pv_lightImpDist, , );
    rtBuffer<SurfaceLightDescriptor> pv_surfaceLightDescriptors;
    
    RT_FUNCTION bool testVisibility(const SurfacePoint &shadingSurfacePoint, const SurfacePoint &lightSurfacePoint, 
                                      Vector3D* shadowRayDir, float* squaredDistance, float* fractionalVisibility) {
        VLRAssert(shadingSurfacePoint.atInfinity == false, "Shading point must be in finite region.");

        *shadowRayDir = lightSurfacePoint.calcDirectionFrom(shadingSurfacePoint.position, squaredDistance);
        optix::Ray shadowRay(asOptiXType(shadingSurfacePoint.position), asOptiXType(*shadowRayDir), RayType::Shadow, 1e-4f);
        if (lightSurfacePoint.atInfinity)
            shadowRay.tmax = FLT_MAX;
        else
            shadowRay.tmax = std::sqrt(*squaredDistance) * 0.9999f;

        ShadowRayPayload shadowPayload;
        rtTrace(pv_topGroup, shadowRay, shadowPayload);

        *fractionalVisibility = shadowPayload.fractionalVisibility;

        return *fractionalVisibility > 0;
    }

    RT_FUNCTION void selectSurfaceLight(float lightSample, SurfaceLight* light, float* lightProb, float* remapped) {
        uint32_t lightIdx = pv_lightImpDist.sample(lightSample, lightProb, remapped);
        *light = SurfaceLight(pv_surfaceLightDescriptors[lightIdx]);
    }



    // JP: �@���}�b�v�ɏ]���ăV�F�[�f�B���O�t���[����ύX����B
    // EN: perturb the shading frame according to the normal map.
    RT_FUNCTION void applyBumpMapping(const Normal3D &normalLocal, SurfacePoint* surfPt) {
        const ReferenceFrame &originalFrame = surfPt->shadingFrame;

        Vector3D nLocal = normalLocal;
        Vector3D tLocal = Vector3D::Ex() - dot(nLocal, Vector3D::Ex()) * nLocal;
        Vector3D bLocal = Vector3D::Ey() - dot(nLocal, Vector3D::Ey()) * nLocal;
        Vector3D t = normalize(originalFrame.fromLocal(tLocal));
        Vector3D b = normalize(originalFrame.fromLocal(bLocal));
        Vector3D n = normalize(originalFrame.fromLocal(nLocal));
        ReferenceFrame bumpFrame(t, b, n);

        surfPt->shadingFrame = bumpFrame;
    }



    rtDeclareVariable(ThinLensCamera, pv_thinLensCamera, , );
    
    RT_FUNCTION RGBSpectrum sampleLensPosition(const LensPosSample &sample, LensPosQueryResult* result) {
        Matrix3x3 rotMat = pv_thinLensCamera.orientation.toMatrix3x3();

        float lx, ly;
        concentricSampleDisk(sample.uPos[0], sample.uPos[1], &lx, &ly);
        Point3D orgLocal = Point3D(pv_thinLensCamera.lensRadius * lx, pv_thinLensCamera.lensRadius * ly, 0.0f);

        Normal3D geometricNormal = normalize(rotMat * Normal3D(0, 0, 1));

        ReferenceFrame shadingFrame;
        shadingFrame.z = (Vector3D)geometricNormal;
        shadingFrame.x = normalize(rotMat * Vector3D(1, 0, 0));
        shadingFrame.y = cross(shadingFrame.z, shadingFrame.x);

        SurfacePoint surfPt;
        surfPt.position = rotMat * orgLocal + pv_thinLensCamera.position;
        surfPt.shadingFrame = shadingFrame;
        surfPt.atInfinity = false;

        surfPt.geometricNormal = geometricNormal;
        surfPt.u = lx;
        surfPt.v = ly;
        surfPt.texCoord = TexCoord2D::Zero();
        //surfPt.tc0Direction = Vector3D::Zero();

        result->surfPt = surfPt;
        result->areaPDF = pv_thinLensCamera.lensRadius > 0.0f ? 1.0f / (M_PIf * pv_thinLensCamera.lensRadius * pv_thinLensCamera.lensRadius) : 1.0f;
        result->posType = pv_thinLensCamera.lensRadius > 0.0f ? DirectionType::LowFreq() : DirectionType::Delta0D();

        return RGBSpectrum::One();
    }

    RT_FUNCTION RGBSpectrum sampleIDF(const SurfacePoint &surfPt, const IDFSample &sample, IDFQueryResult* result) {
        Point3D orgLocal = Point3D(pv_thinLensCamera.lensRadius * surfPt.u, pv_thinLensCamera.lensRadius * surfPt.v, 0.0f);

        Point3D pFocus = Point3D(pv_thinLensCamera.opWidth * (0.5f - sample.uDir[0]),
                                 pv_thinLensCamera.opHeight * (0.5f - sample.uDir[1]),
                                 pv_thinLensCamera.objPlaneDistance);

        Vector3D dirLocal = normalize(pFocus - orgLocal);
        result->dirLocal = dirLocal;
        result->dirPDF = pv_thinLensCamera.imgPlaneDistance * pv_thinLensCamera.imgPlaneDistance / ((dirLocal.z * dirLocal.z * dirLocal.z) * pv_thinLensCamera.imgPlaneArea);
        result->sampledType = DirectionType::Acquisition() | DirectionType::LowFreq();

        return RGBSpectrum::One();
    }



    rtDeclareVariable(optix::uint2, sm_launchIndex, rtLaunchIndex, );
    rtDeclareVariable(optix::Ray, sm_ray, rtCurrentRay, );
    rtDeclareVariable(Payload, sm_payload, rtPayload, );
    rtDeclareVariable(ShadowRayPayload, sm_shadowRayPayload, rtPayload, );
    rtDeclareVariable(HitPointParameter, a_hitPointParam, attribute hitPointParam, );

    rtBuffer<PCG32RNG, 2> pv_rngBuffer;
    rtBuffer<RGBSpectrum, 2> pv_outputBuffer;

    // per GeometryInstance
    rtDeclareVariable(progSigDecodeTexCoord, pv_progDecodeTexCoord, , );
    rtDeclareVariable(progSigDecodeHitPoint, pv_progDecodeHitPoint, , );

    // per Material
    rtDeclareVariable(progSigFetchAlpha, pv_progFetchAlpha, , );
    rtDeclareVariable(progSigFetchNormal, pv_progFetchNormal, , );

    // Common Any Hit Program for All Primitive Types and Materials for non-shadow rays
    RT_PROGRAM void stochasticAlphaAnyHit() {
        HitPointParameter hitPointParam = a_hitPointParam;
        TexCoord2D texCoord = pv_progDecodeTexCoord(hitPointParam);

        float alpha = pv_progFetchAlpha(texCoord);

        PCG32RNG &rng = pv_rngBuffer[sm_launchIndex];
        if (rng.getFloat0cTo1o() >= alpha)
            rtIgnoreIntersection();
    }

    // Common Any Hit Program for All Primitive Types and Materials for shadow rays
    RT_PROGRAM void alphaAnyHit() {
        HitPointParameter hitPointParam = a_hitPointParam;
        TexCoord2D texCoord = pv_progDecodeTexCoord(hitPointParam);

        float alpha = pv_progFetchAlpha(texCoord);

        sm_shadowRayPayload.fractionalVisibility *= alpha;
        if (sm_shadowRayPayload.fractionalVisibility == 0.0f)
            rtTerminateRay();
    }

    // Common Closest Hit Program for All Primitive Types and Materials
    RT_PROGRAM void pathTracingIteration() {
        PCG32RNG &rng = pv_rngBuffer[sm_launchIndex];

        SurfacePoint surfPt;
        HitPointParameter hitPointParam = a_hitPointParam;
        pv_progDecodeHitPoint(hitPointParam, &surfPt);

        applyBumpMapping(pv_progFetchNormal(surfPt.texCoord), &surfPt);

        Vector3D dirOutLocal = surfPt.shadingFrame.toLocal(-asVector3D(sm_ray.direction));

        // implicit light sampling
        /*if (surfPt.isEmitting())*/ {
            float bsdfPDF = sm_payload.prevDirPDF;

            RGBSpectrum Le = evaluateEmittance(surfPt.texCoord) * evaluateEDF(surfPt.texCoord, EDFQuery(), dirOutLocal);
            float dist2 = surfPt.calcSquaredDistance(asPoint3D(sm_ray.origin));
            float lightPDF = 1.0f;// = si.getLightProb() * surfPt.evaluateAreaPDF() * dist2 / surfPt.calcCosTerm(asVector3D(ray.direction));

            float MISWeight = 1.0f;
            if (!sm_payload.prevSampledType.isDelta() && sm_ray.ray_type != RayType::Primary)
                MISWeight = (bsdfPDF * bsdfPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);

            sm_payload.contribution += sm_payload.alpha * Le * MISWeight;
        }
        if (surfPt.atInfinity) {
            sm_payload.terminate = true;
            return;
        }

        // Russian roulette
        float continueProb = std::min(sm_payload.alpha.importance() / sm_payload.initY, 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb) {
            sm_payload.terminate = true;
            return;
        }

        Normal3D geomNormalLocal = surfPt.shadingFrame.toLocal(surfPt.geometricNormal);
        BSDFQuery fsQuery(dirOutLocal, geomNormalLocal, DirectionType::All());

        // Next Event Estimation (explicit light sampling)
        /*if (bsdf->hasNonDelta())*/ {
            float lightSample = rng.getFloat0cTo1o();
            SurfaceLight light;
            float lightProb;
            selectSurfaceLight(lightSample, &light, &lightProb, &lightSample);

            SurfaceLightPosSample lpSample(lightSample, rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
            SurfaceLightPosQueryResult lpResult;
            RGBSpectrum M = light.sample(lpSample, &lpResult);

            Vector3D shadowRayDir;
            float squaredDistance;
            float fractionalVisibility;
            if (testVisibility(surfPt, lpResult.surfPt, &shadowRayDir, &squaredDistance, &fractionalVisibility)) {
                Vector3D shadowRayDir_l = lpResult.surfPt.toLocal(-shadowRayDir);
                Vector3D shadowRayDir_sn = surfPt.toLocal(shadowRayDir);

                RGBSpectrum Le = M * evaluateEDF(lpResult.surfPt.texCoord, EDFQuery(), shadowRayDir_l);
                float lightPDF = lightProb * lpResult.areaPDF;

                RGBSpectrum fs = evaluateBSDF(surfPt.texCoord, fsQuery, shadowRayDir_sn);
                float cosLight = lpResult.surfPt.calcCosTerm(-shadowRayDir);
                float bsdfPDF = evaluateBSDF_PDF(surfPt.texCoord, fsQuery, shadowRayDir_sn) * cosLight / squaredDistance;

                float MISWeight = 1.0f;
                if (!lpResult.posType.isDelta() && !std::isinf(lpResult.areaPDF))
                    MISWeight = (lightPDF * lightPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);

                float G = fractionalVisibility * absDot(shadowRayDir_sn, geomNormalLocal) * cosLight / squaredDistance;
                sm_payload.contribution += sm_payload.alpha * Le * fs * (G * MISWeight / lightPDF);
            }
        }

        BSDFSample sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        BSDFQueryResult fsResult;
        RGBSpectrum fs = sampleBSDF(surfPt.texCoord, fsQuery, sample, &fsResult);
        if (fs == RGBSpectrum::Zero() || fsResult.dirPDF == 0.0f) {
            sm_payload.terminate = true;
            return;
        }

        sm_payload.alpha *= fs * absDot(fsResult.dirLocal, geomNormalLocal) / fsResult.dirPDF;

        Vector3D dirIn = surfPt.fromLocal(fsResult.dirLocal);
        sm_payload.origin = surfPt.position;
        sm_payload.direction = dirIn;
        sm_payload.prevDirPDF = fsResult.dirPDF;
        sm_payload.prevSampledType = fsResult.sampledType;
        sm_payload.terminate = false;
    }

    // Ray Generation Program
    RT_PROGRAM void pathTracing() {
        PCG32RNG &rng = pv_rngBuffer[sm_launchIndex];

        optix::float2 p = make_float2(sm_launchIndex.x + rng.getFloat0cTo1o(), sm_launchIndex.y + rng.getFloat0cTo1o());
        optix::size_t2 bufferSize = pv_outputBuffer.size();
        optix::uint2 imageSize = make_uint2(bufferSize.x, bufferSize.y);

        LensPosSample We0Sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        LensPosQueryResult We0Result;
        RGBSpectrum We0 = sampleLensPosition(We0Sample, &We0Result);

        IDFSample We1Sample(p.x / imageSize.x, p.y / imageSize.y);
        IDFQueryResult We1Result;
        RGBSpectrum We1 = sampleIDF(We0Result.surfPt, We1Sample, &We1Result);

        Vector3D rayDir = We0Result.surfPt.fromLocal(We1Result.dirLocal);
        RGBSpectrum alpha = (We0 * We1) * (We0Result.surfPt.calcCosTerm(rayDir) / (We0Result.areaPDF * We1Result.dirPDF));

        optix::Ray ray = optix::make_Ray(asOptiXType(We0Result.surfPt.position), asOptiXType(rayDir), RayType::Primary, 0.0f, INFINITY);
        Payload payload;
        payload.initY = alpha.importance();
        payload.alpha = alpha;
        payload.contribution = RGBSpectrum::Zero();

        while (true) {
            rtTrace(pv_topGroup, ray, payload);

            if (payload.terminate)
                break;

            ray = optix::make_Ray(asOptiXType(payload.origin), asOptiXType(payload.direction), RayType::Scattered, 1e-4f, INFINITY);
        }

        RGBSpectrum &contribution = pv_outputBuffer[sm_launchIndex];
        contribution += payload.contribution;
    }
}
