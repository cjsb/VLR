#include "hip/hip_runtime.h"
﻿#include "kernel_common.cuh"

namespace VLR {
    RT_FUNCTION DirectionType sideTest(const Normal3D &ng, const Vector3D &d0, const Vector3D &d1) {
        bool reflect = dot(Vector3D(ng), d0) * dot(Vector3D(ng), d1) > 0;
        return DirectionType::AllFreq() | (reflect ? DirectionType::Reflection() : DirectionType::Transmission());
    }



    class FresnelConductor {
        RGBSpectrum m_eta;
        RGBSpectrum m_k;

    public:
        RT_FUNCTION FresnelConductor(const RGBSpectrum &eta, const RGBSpectrum &k) : m_eta(eta), m_k(k) {}

        RT_FUNCTION RGBSpectrum evaluate(float cosEnter) const {
            cosEnter = std::fabs(cosEnter);
            float cosEnter2 = cosEnter * cosEnter;
            RGBSpectrum _2EtaCosEnter = 2.0f * m_eta * cosEnter;
            RGBSpectrum tmp_f = m_eta * m_eta + m_k * m_k;
            RGBSpectrum tmp = tmp_f * cosEnter2;
            RGBSpectrum Rparl2 = (tmp - _2EtaCosEnter + 1) / (tmp + _2EtaCosEnter + 1);
            RGBSpectrum Rperp2 = (tmp_f - _2EtaCosEnter + cosEnter2) / (tmp_f + _2EtaCosEnter + cosEnter2);

            return (Rparl2 + Rperp2) / 2.0f;
        }
        RT_FUNCTION float evaluate(float cosEnter, uint32_t wlIdx) const {
            cosEnter = std::fabs(cosEnter);
            float cosEnter2 = cosEnter * cosEnter;
            float _2EtaCosEnter = 2.0f * m_eta[wlIdx] * cosEnter;
            float tmp_f = m_eta[wlIdx] * m_eta[wlIdx] + m_k[wlIdx] * m_k[wlIdx];
            float tmp = tmp_f * cosEnter2;
            float Rparl2 = (tmp - _2EtaCosEnter + 1) / (tmp + _2EtaCosEnter + 1);
            float Rperp2 = (tmp_f - _2EtaCosEnter + cosEnter2) / (tmp_f + _2EtaCosEnter + cosEnter2);

            return (Rparl2 + Rperp2) / 2.0f;
        }
    };



    class FresnelDielectric {
        RGBSpectrum m_etaExt;
        RGBSpectrum m_etaInt;

    public:
        RT_FUNCTION FresnelDielectric(const RGBSpectrum &etaExt, const RGBSpectrum &etaInt) : m_etaExt(etaExt), m_etaInt(etaInt) {}

        RT_FUNCTION RGBSpectrum etaExt() const { return m_etaExt; }
        RT_FUNCTION RGBSpectrum etaInt() const { return m_etaInt; }

        RT_FUNCTION RGBSpectrum evaluate(float cosEnter) const {
            cosEnter = clamp(cosEnter, -1.0f, 1.0f);

            bool entering = cosEnter > 0.0f;
            const RGBSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
            const RGBSpectrum &eExit = entering ? m_etaInt : m_etaExt;

            RGBSpectrum sinExit = eEnter / eExit * std::sqrt(std::fmax(0.0f, 1.0f - cosEnter * cosEnter));
            RGBSpectrum ret = RGBSpectrum::Zero();
            cosEnter = std::fabs(cosEnter);
            for (int i = 0; i < RGBSpectrum::NumComponents(); ++i) {
                if (sinExit[i] >= 1.0f) {
                    ret[i] = 1.0f;
                }
                else {
                    float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit[i] * sinExit[i]));
                    ret[i] = evalF(eEnter[i], eExit[i], cosEnter, cosExit);
                }
            }
            return ret;
        }
        RT_FUNCTION float evaluate(float cosEnter, uint32_t wlIdx) const {
            cosEnter = clamp(cosEnter, -1.0f, 1.0f);

            bool entering = cosEnter > 0.0f;
            const float &eEnter = entering ? m_etaExt[wlIdx] : m_etaInt[wlIdx];
            const float &eExit = entering ? m_etaInt[wlIdx] : m_etaExt[wlIdx];

            float sinExit = eEnter / eExit * std::sqrt(std::fmax(0.0f, 1.0f - cosEnter * cosEnter));
            cosEnter = std::fabs(cosEnter);
            if (sinExit >= 1.0f) {
                return 1.0f;
            }
            else {
                float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit * sinExit));
                return evalF(eEnter, eExit, cosEnter, cosExit);
            }
        }

        RT_FUNCTION static float evalF(float etaEnter, float etaExit, float cosEnter, float cosExit);
    };

    RT_FUNCTION float FresnelDielectric::evalF(float etaEnter, float etaExit, float cosEnter, float cosExit) {
        float Rparl = ((etaExit * cosEnter) - (etaEnter * cosExit)) / ((etaExit * cosEnter) + (etaEnter * cosExit));
        float Rperp = ((etaEnter * cosEnter) - (etaExit * cosExit)) / ((etaEnter * cosEnter) + (etaExit * cosExit));
        return (Rparl * Rparl + Rperp * Rperp) / 2.0f;
    }



    class GGXMicrofacetDistribution {
        float m_alpha_gx;
        float m_alpha_gy;

    public:
        RT_FUNCTION GGXMicrofacetDistribution(float alpha_gx, float alpha_gy) :
            m_alpha_gx(alpha_gx), m_alpha_gy(alpha_gy) {}

        RT_FUNCTION float evaluate(const Normal3D &m) {
            if (m.z <= 0)
                return 0.0f;
            float temp = m.x * m.x / (m_alpha_gx * m_alpha_gx) + m.y * m.y / (m_alpha_gy * m_alpha_gy) + m.z * m.z;
            return 1.0f / (M_PIf * m_alpha_gx * m_alpha_gy * temp * temp);
        }

        RT_FUNCTION float evaluateSmithG1(const Vector3D &v, const Normal3D &m) {
            float chi = (dot(v, m) / v.z) > 0 ? 1 : 0;
            float tanTheta_v_alpha_go_2 = (v.x * v.x * m_alpha_gx * m_alpha_gx + v.y * v.y * m_alpha_gy * m_alpha_gy) / (v.z * v.z);
            return chi * 2 / (1 + std::sqrt(1 + tanTheta_v_alpha_go_2));
        }

        RT_FUNCTION float sample(const Vector3D &v, float u0, float u1, Normal3D* m, float* normalPDF) {
            // stretch view
            Vector3D sv = normalize(Vector3D(m_alpha_gx * v.x, m_alpha_gy * v.y, v.z));

            // orthonormal basis
            //        Vector3D T1 = (sv.z < 0.9999f) ? normalize(cross(sv, Vector3D::Ez)) : Vector3D::Ex;
            //        Vector3D T2 = cross(T1, sv);
            float distIn2D = std::sqrt(sv.x * sv.x + sv.y * sv.y);
            float recDistIn2D = 1.0f / distIn2D;
            Vector3D T1 = (sv.z < 0.9999f) ? Vector3D(sv.y * recDistIn2D, -sv.x * recDistIn2D, 0) : Vector3D::Ex();
            Vector3D T2 = Vector3D(T1.y * sv.z, -T1.x * sv.z, distIn2D);

            // sample point with polar coordinates (r, phi)
            float a = 1.0f / (1.0f + sv.z);
            float r = std::sqrt(u0);
            float phi = M_PIf * ((u1 < a) ? u1 / a : 1 + (u1 - a) / (1.0f - a));
            float P1 = r * std::cos(phi);
            float P2 = r * std::sin(phi) * ((u1 < a) ? 1.0 : sv.z);

            // compute normal
            *m = P1 * T1 + P2 * T2 + std::sqrt(1.0f - P1 * P1 - P2 * P2) * sv;

            // unstretch
            *m = normalize(Normal3D(m_alpha_gx * m->x, m_alpha_gy * m->y, m->z));

            float D = evaluate(*m);
            *normalPDF = evaluateSmithG1(v, *m) * absDot(v, *m) * D / std::abs(v.z);

            return D;
        }

        RT_FUNCTION float evaluatePDF(const Vector3D &v, const Normal3D &m) {
            return evaluateSmithG1(v, m) * absDot(v, m) * evaluate(m) / std::abs(v.z);
        }
    };



    // ----------------------------------------------------------------
    // Texture Mappings

    RT_CALLABLE_PROGRAM Point3D OffsetAndScaleUVTextureMap2D_map(const uint32_t* texMapDesc, const SurfacePoint &surfPt) {
        const OffsetAndScaleUVTextureMap2D &texMap = *(const OffsetAndScaleUVTextureMap2D*)texMapDesc;
        return Point3D(texMap.scale[0] * surfPt.texCoord.u + texMap.offset[0],
                       texMap.scale[1] * surfPt.texCoord.v + texMap.offset[1],
                       0.0f);
    }

    RT_FUNCTION Point3D textureMap(uint32_t texMapIndex, const SurfacePoint &surfPt) {
        const TextureMapDescriptor &texMapDesc = pv_textureMapDescriptorBuffer[texMapIndex];
        progSigTextureMap progTexMap = (progSigTextureMap)texMapDesc.progTextureMap;
        return progTexMap(texMapDesc.data, surfPt);
    }

    // END: Texture Mappings
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // NullBSDF

    RT_CALLABLE_PROGRAM uint32_t NullBSDF_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        return 0;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullBSDF_getBaseColor(const uint32_t* params) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM bool NullBSDF_matches(const uint32_t* params, DirectionType flags) {
        return false;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullBSDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullBSDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float NullBSDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float NullBSDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        return 0.0f;
    }

    // END: NullBSDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // MatteBRDF

    struct MatteBRDF {
        RGBSpectrum albedo;
        float roughness;
    };

    RT_CALLABLE_PROGRAM uint32_t MatteSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        MatteBRDF &p = *(MatteBRDF*)params;
        const MatteSurfaceMaterial &mat = *(const MatteSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        Point3D texCoord = textureMap(mat.texMap, surfPt);

        optix::float4 texValue = optix::rtTex2D<optix::float4>(mat.texAlbedoRoughness, texCoord.x, texCoord.y);
        p.albedo = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        p.roughness = texValue.w;

        return sizeof(MatteBRDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MatteBRDF_getBaseColor(const uint32_t* params) {
        MatteBRDF &p = *(MatteBRDF*)params;

        return p.albedo;
    }

    RT_CALLABLE_PROGRAM bool MatteBRDF_matches(const uint32_t* params, DirectionType flags) {
        DirectionType m_type = DirectionType::Reflection() | DirectionType::LowFreq();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MatteBRDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        MatteBRDF &p = *(MatteBRDF*)params;

        result->dirLocal = cosineSampleHemisphere(uDir[0], uDir[1]);
        result->dirPDF = result->dirLocal.z / M_PIf;
        result->sampledType = DirectionType::Reflection() | DirectionType::LowFreq();
        result->dirLocal.z *= query.dirLocal.z > 0 ? 1 : -1;

        return p.albedo / M_PIf;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MatteBRDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        MatteBRDF &p = *(MatteBRDF*)params;

        if (query.dirLocal.z * dirLocal.z <= 0.0f) {
            RGBSpectrum fs = RGBSpectrum::Zero();
            return fs;
        }
        RGBSpectrum fs = p.albedo / M_PIf;

        return fs;
    }

    RT_CALLABLE_PROGRAM float MatteBRDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        if (query.dirLocal.z * dirLocal.z <= 0.0f) {
            return 0.0f;
        }

        return std::abs(dirLocal.z) / M_PIf;
    }

    RT_CALLABLE_PROGRAM float MatteBRDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        MatteBRDF &p = *(MatteBRDF*)params;
        return p.albedo.importance(query.wlHint);
    }

    // END: MatteBRDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // SpecularBRDF

    struct SpecularBRDF {
        RGBSpectrum coeffR;
        RGBSpectrum eta;
        RGBSpectrum k;
    };

    RT_CALLABLE_PROGRAM uint32_t SpecularReflectionSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        SpecularBRDF &p = *(SpecularBRDF*)params;
        const SpecularReflectionSurfaceMaterial &mat = *(const SpecularReflectionSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        Point3D texCoord = textureMap(mat.texMap, surfPt);

        optix::float4 texValue;
        texValue = optix::rtTex2D<optix::float4>(mat.texCoeffR, texCoord.x, texCoord.y);
        p.coeffR = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.texEta, texCoord.x, texCoord.y);
        p.eta = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.tex_k, texCoord.x, texCoord.y);
        p.k = RGBSpectrum(texValue.x, texValue.y, texValue.z);

        return sizeof(SpecularBRDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBRDF_getBaseColor(const uint32_t* params) {
        SpecularBRDF &p = *(SpecularBRDF*)params;

        return p.coeffR;
    }

    RT_CALLABLE_PROGRAM bool SpecularBRDF_matches(const uint32_t* params, DirectionType flags) {
        DirectionType m_type = DirectionType::Reflection() | DirectionType::Delta0D();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBRDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        SpecularBRDF &p = *(SpecularBRDF*)params;

        FresnelConductor fresnel(p.eta, p.k);

        result->dirLocal = Vector3D(-query.dirLocal.x, -query.dirLocal.y, query.dirLocal.z);
        result->dirPDF = 1.0f;
        result->sampledType = DirectionType::Reflection() | DirectionType::Delta0D();
        RGBSpectrum fs = p.coeffR * fresnel.evaluate(query.dirLocal.z) / std::fabs(query.dirLocal.z);

        return fs;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBRDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float SpecularBRDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float SpecularBRDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        SpecularBRDF &p = *(SpecularBRDF*)params;

        FresnelDielectric fresnel(p.eta, p.k);

        return (p.coeffR * fresnel.evaluate(query.dirLocal.z)).importance(query.wlHint);
    }

    // END: SpecularBRDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // SpecularBSDF

    struct SpecularBSDF {
        RGBSpectrum coeff;
        RGBSpectrum etaExt;
        RGBSpectrum etaInt;
        bool dispersive;
    };

    RT_CALLABLE_PROGRAM uint32_t SpecularScatteringSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        SpecularBSDF &p = *(SpecularBSDF*)params;
        const SpecularScatteringSurfaceMaterial &mat = *(const SpecularScatteringSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        Point3D texCoord = textureMap(mat.texMap, surfPt);

        optix::float4 texValue;
        texValue = optix::rtTex2D<optix::float4>(mat.texCoeff, texCoord.x, texCoord.y);
        p.coeff = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.texEtaExt, texCoord.x, texCoord.y);
        p.etaExt = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.texEtaInt, texCoord.x, texCoord.y);
        p.etaInt = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        p.dispersive = !wavelengthSelected;

        return sizeof(SpecularBSDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBSDF_getBaseColor(const uint32_t* params) {
        SpecularBSDF &p = *(SpecularBSDF*)params;

        return p.coeff;
    }

    RT_CALLABLE_PROGRAM bool SpecularBSDF_matches(const uint32_t* params, DirectionType flags) {
        DirectionType m_type = DirectionType::WholeSphere() | DirectionType::Delta0D();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBSDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        SpecularBSDF &p = *(SpecularBSDF*)params;

        bool entering = query.dirLocal.z >= 0.0f;

        const RGBSpectrum &eEnter = entering ? p.etaExt : p.etaInt;
        const RGBSpectrum &eExit = entering ? p.etaInt : p.etaExt;
        FresnelDielectric fresnel(eEnter, eExit);

        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

        RGBSpectrum F = fresnel.evaluate(dirV.z);
        float reflectProb = F.importance(query.wlHint);
        if (query.dirTypeFilter.isReflection())
            reflectProb = 1.0f;
        if (query.dirTypeFilter.isTransmission())
            reflectProb = 0.0f;
        if (uComponent < reflectProb) {
            if (dirV.z == 0.0f) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            Vector3D dirL = Vector3D(-dirV.x, -dirV.y, dirV.z);
            result->dirLocal = entering ? dirL : -dirL;
            result->dirPDF = reflectProb;
            result->sampledType = DirectionType::Reflection() | DirectionType::Delta0D();
            RGBSpectrum fs = p.coeff * F / std::fabs(dirV.z);

            return fs;
        }
        else {
            float sinEnter2 = 1.0f - dirV.z * dirV.z;
            float recRelIOR = eEnter[query.wlHint] / eExit[query.wlHint];// reciprocal of relative IOR.
            float sinExit2 = recRelIOR * recRelIOR * sinEnter2;

            if (sinExit2 >= 1.0f) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit2));
            Vector3D dirL = Vector3D(recRelIOR * -dirV.x, recRelIOR * -dirV.y, -cosExit);
            result->dirLocal = entering ? dirL : -dirL;
            result->dirPDF = 1.0f - reflectProb;
            result->sampledType = DirectionType::Transmission() | DirectionType::Delta0D() | (p.dispersive ? DirectionType::Dispersive() : DirectionType());

            RGBSpectrum ret = RGBSpectrum::Zero();
            ret[query.wlHint] = p.coeff[query.wlHint] * (1.0f - F[query.wlHint]);
            RGBSpectrum fs = ret / std::fabs(cosExit);

            return fs;
        }
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBSDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float SpecularBSDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float SpecularBSDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        SpecularBSDF &p = *(SpecularBSDF*)params;
        return p.coeff.importance(query.wlHint);
    }

    // END: SpecularBSDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // MicrofacetBRDF

    struct MicrofacetBRDF {
        RGBSpectrum eta;
        RGBSpectrum k;
        float roughnessX;
        float roughnessY;
    };

    RT_CALLABLE_PROGRAM uint32_t MicrofacetReflectionSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        MicrofacetBRDF &p = *(MicrofacetBRDF*)params;
        const MicrofacetReflectionSurfaceMaterial &mat = *(const MicrofacetReflectionSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        Point3D texCoord = textureMap(mat.texMap, surfPt);

        optix::float4 texValue;
        optix::float2 texValueF2;
        texValue = optix::rtTex2D<optix::float4>(mat.texEta, texCoord.x, texCoord.y);
        p.eta = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.tex_k, texCoord.x, texCoord.y);
        p.k = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValueF2 = optix::rtTex2D<optix::float2>(mat.texRoughness, texCoord.x, texCoord.y);
        p.roughnessX = texValueF2.x;
        p.roughnessY = texValueF2.y;

        return sizeof(MicrofacetBRDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MicrofacetBRDF_getBaseColor(const uint32_t* params) {
        MicrofacetBRDF &p = *(MicrofacetBRDF*)params;

        FresnelDielectric fresnel(p.eta, p.k);

        return fresnel.evaluate(1.0f);
    }

    RT_CALLABLE_PROGRAM bool MicrofacetBRDF_matches(const uint32_t* params, DirectionType flags) {
        DirectionType m_type = DirectionType::Reflection() | DirectionType::HighFreq();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MicrofacetBRDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        MicrofacetBRDF &p = *(MicrofacetBRDF*)params;

        bool entering = query.dirLocal.z >= 0.0f;

        FresnelDielectric fresnel(p.eta, p.k);

        float alphaX = p.roughnessX * p.roughnessX;
        float alphaY = p.roughnessY * p.roughnessY;
        GGXMicrofacetDistribution ggx(alphaX, alphaY);

        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

        // JP: ハーフベクトルをサンプルして、最終的な方向サンプルを生成する。
        // EN: sample a half vector, then generate a resulting direction sample based on it.
        Normal3D m;
        float mPDF;
        float D = ggx.sample(dirV, uDir[0], uDir[1], &m, &mPDF);
        float dotHV = dot(dirV, m);
        if (dotHV <= 0) {
            result->dirPDF = 0.0f;
            return RGBSpectrum::Zero();
        }

        Vector3D dirL = 2 * dotHV * m - dirV;
        result->dirLocal = entering ? dirL : -dirL;
        if (dirL.z * dirV.z <= 0) {
            result->dirPDF = 0.0f;
            return RGBSpectrum::Zero();
        }

        float commonPDFTerm = 1.0f / (4 * dotHV);
        result->dirPDF = commonPDFTerm * mPDF;
        result->sampledType = DirectionType::Reflection() | DirectionType::HighFreq();

        RGBSpectrum F = fresnel.evaluate(dotHV);
        float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
        RGBSpectrum fs = F * D * G / (4 * dirV.z * dirL.z);

        //VLRAssert(fs.allFinite(), "fs: %s, F: %s, G, %g, D: %g, wlIdx: %u, qDir: %s, rDir: %s",
        //          fs.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

        return fs;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MicrofacetBRDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        MicrofacetBRDF &p = *(MicrofacetBRDF*)params;

        bool entering = query.dirLocal.z >= 0.0f;

        FresnelConductor fresnel(p.eta, p.k);

        float alphaX = p.roughnessX * p.roughnessX;
        float alphaY = p.roughnessY * p.roughnessY;
        GGXMicrofacetDistribution ggx(alphaX, alphaY);

        Vector3D dirL = entering ? dirLocal : -dirLocal;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
        float dotNVdotNL = dirL.z * dirV.z;

        if (dotNVdotNL <= 0)
            return RGBSpectrum::Zero();

        Normal3D m = halfVector(dirV, dirL);
        float dotHV = dot(dirV, m);
        float D = ggx.evaluate(m);

        RGBSpectrum F = fresnel.evaluate(dotHV);
        float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
        RGBSpectrum fs = F * D * G / (4 * dotNVdotNL);

        //VLRAssert(fs.allFinite(), "fs: %s, F: %s, G, %g, D: %g, wlIdx: %u, qDir: %s, dir: %s",
        //          fs.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

        return fs;
    }

    RT_CALLABLE_PROGRAM float MicrofacetBRDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        MicrofacetBRDF &p = *(MicrofacetBRDF*)params;

        bool entering = query.dirLocal.z >= 0.0f;

        FresnelConductor fresnel(p.eta, p.k);

        float alphaX = p.roughnessX * p.roughnessX;
        float alphaY = p.roughnessY * p.roughnessY;
        GGXMicrofacetDistribution ggx(alphaX, alphaY);

        Vector3D dirL = entering ? dirLocal : -dirLocal;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
        float dotNVdotNL = dirL.z * dirV.z;

        if (dotNVdotNL <= 0.0f)
            return 0.0f;

        Normal3D m = halfVector(dirV, dirL);
        float dotHV = dot(dirV, m);
        if (dotHV <= 0)
            return 0.0f;

        float mPDF = ggx.evaluatePDF(dirV, m);
        float commonPDFTerm = 1.0f / (4 * dotHV);
        float ret = commonPDFTerm * mPDF;

        //VLRAssert(std::isfinite(commonPDFTerm) && std::isfinite(mPDF),
        //          "commonPDFTerm: %g, mPDF: %g, wlIdx: %u, qDir: %s, dir: %s",
        //          commonPDFTerm, mPDF, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

        return ret;
    }

    RT_CALLABLE_PROGRAM float MicrofacetBRDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        MicrofacetBRDF &p = *(MicrofacetBRDF*)params;

        FresnelDielectric fresnel(p.eta, p.k);

        float expectedDotHV = query.dirLocal.z;

        return fresnel.evaluate(expectedDotHV).importance(query.wlHint);
    }

    // END: MicrofacetBRDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // MicrofacetBSDF

    struct MicrofacetBSDF {
        RGBSpectrum coeff;
        RGBSpectrum etaExt;
        RGBSpectrum etaInt;
        float roughnessX;
        float roughnessY;
    };

    RT_CALLABLE_PROGRAM uint32_t MicrofacetScatteringSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        MicrofacetBSDF &p = *(MicrofacetBSDF*)params;
        const MicrofacetScatteringSurfaceMaterial &mat = *(const MicrofacetScatteringSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        Point3D texCoord = textureMap(mat.texMap, surfPt);

        optix::float4 texValue;
        optix::float2 texValueF2;
        texValue = optix::rtTex2D<optix::float4>(mat.texCoeff, texCoord.x, texCoord.y);
        p.coeff = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.texEtaExt, texCoord.x, texCoord.y);
        p.etaExt = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.texEtaInt, texCoord.x, texCoord.y);
        p.etaInt = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValueF2 = optix::rtTex2D<optix::float2>(mat.texRoughness, texCoord.x, texCoord.y);
        p.roughnessX = texValueF2.x;
        p.roughnessY = texValueF2.y;

        return sizeof(MicrofacetBSDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MicrofacetBSDF_getBaseColor(const uint32_t* params) {
        MicrofacetBSDF &p = *(MicrofacetBSDF*)params;

        return p.coeff;
    }

    RT_CALLABLE_PROGRAM bool MicrofacetBSDF_matches(const uint32_t* params, DirectionType flags) {
        DirectionType m_type = DirectionType::WholeSphere() | DirectionType::HighFreq();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MicrofacetBSDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        MicrofacetBSDF &p = *(MicrofacetBSDF*)params;

        bool entering = query.dirLocal.z >= 0.0f;

        const RGBSpectrum &eEnter = entering ? p.etaExt : p.etaInt;
        const RGBSpectrum &eExit = entering ? p.etaInt : p.etaExt;
        FresnelDielectric fresnel(eEnter, eExit);

        float alphaX = p.roughnessX * p.roughnessX;
        float alphaY = p.roughnessY * p.roughnessY;
        GGXMicrofacetDistribution ggx(alphaX, alphaY);

        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

        // JP: ハーフベクトルをサンプルする。
        // EN: sample a half vector.
        Normal3D m;
        float mPDF;
        float D = ggx.sample(dirV, uDir[0], uDir[1], &m, &mPDF);
        float dotHV = dot(dirV, m);
        if (dotHV <= 0 || std::isnan(D)) {
            result->dirPDF = 0.0f;
            return RGBSpectrum::Zero();
        }

        // JP: サンプルしたハーフベクトルからフレネル項の値を計算して、反射か透過を選択する。
        // EN: calculate the Fresnel term using the sampled half vector, then select reflection or transmission.
        RGBSpectrum F = fresnel.evaluate(dotHV);
        float reflectProb = F.importance(query.wlHint);
        if (query.dirTypeFilter.isReflection())
            reflectProb = 1.0f;
        if (query.dirTypeFilter.isTransmission())
            reflectProb = 0.0f;
        if (uComponent < reflectProb) {
            // JP: 最終的な方向サンプルを生成する。
            // EN: calculate a resulting direction.
            Vector3D dirL = 2 * dotHV * m - dirV;
            result->dirLocal = entering ? dirL : -dirL;
            if (dirL.z * dirV.z <= 0) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            float commonPDFTerm = reflectProb / (4 * dotHV);
            result->dirPDF = commonPDFTerm * mPDF;
            result->sampledType = DirectionType::Reflection() | DirectionType::HighFreq();

            float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
            RGBSpectrum fs = F * D * G / (4 * dirV.z * dirL.z);

            //VLRAssert(fs.allFinite(), "fs: %s, F: %g, %g, %g, G, %g, D: %g, wlIdx: %u, qDir: (%g, %g, %g), rDir: (%g, %g, %g)",
            //          fs.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, 
            //          dirV.x, dirV.y, dirV.z, dirL.x, dirL.y, dirL.z);

            return fs;
        }
        else {
            // JP: 最終的な方向サンプルを生成する。
            // EN: calculate a resulting direction.
            float recRelIOR = eEnter[query.wlHint] / eExit[query.wlHint];
            float innerRoot = 1 + recRelIOR * recRelIOR * (dotHV * dotHV - 1);
            if (innerRoot < 0) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            Vector3D dirL = (recRelIOR * dotHV - std::sqrt(innerRoot)) * m - recRelIOR * dirV;
            result->dirLocal = entering ? dirL : -dirL;
            if (dirL.z * dirV.z >= 0) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            float dotHL = dot(dirL, m);
            float commonPDFTerm = (1 - reflectProb) / std::pow(eEnter[query.wlHint] * dotHV + eExit[query.wlHint] * dotHL, 2);
            result->dirPDF = commonPDFTerm * mPDF * eExit[query.wlHint] * eExit[query.wlHint] * std::fabs(dotHL);
            result->sampledType = DirectionType::Transmission() | DirectionType::HighFreq();

            // JP: マイクロファセットBSDFの各項の値を波長成分ごとに計算する。
            // EN: calculate the value of each term of the microfacet BSDF for each wavelength component.
            RGBSpectrum ret = RGBSpectrum::Zero();
            for (int wlIdx = 0; wlIdx < RGBSpectrum::NumComponents(); ++wlIdx) {
                Normal3D m_wl = normalize(-(eEnter[wlIdx] * dirV + eExit[wlIdx] * dirL) * (entering ? 1 : -1));
                float dotHV_wl = dot(dirV, m_wl);
                float dotHL_wl = dot(dirL, m_wl);
                float F_wl = fresnel.evaluate(dotHV_wl, wlIdx);
                float G_wl = ggx.evaluateSmithG1(dirV, m_wl) * ggx.evaluateSmithG1(dirL, m_wl);
                float D_wl = ggx.evaluate(m_wl);
                ret[wlIdx] = std::fabs(dotHV_wl * dotHL_wl) * (1 - F_wl) * G_wl * D_wl / std::pow(eEnter[wlIdx] * dotHV_wl + eExit[wlIdx] * dotHL_wl, 2);

                //VLRAssert(std::isfinite(ret[wlIdx]), "fs: %g, F: %g, G, %g, D: %g, wlIdx: %u, qDir: %s",
                //          ret[wlIdx], F_wl, G_wl, D_wl, query.wlHint, dirV.toString().c_str());
            }
            ret /= std::fabs(dirV.z * dirL.z);
            ret *= eEnter * eEnter;
            //ret *= query.adjoint ? (eExit * eExit) : (eEnter * eEnter);// adjoint: need to cancel eEnter^2 / eExit^2 => eEnter^2 * (eExit^2 / eEnter^2)

            //VLRAssert(ret.allFinite(), "fs: %s, wlIdx: %u, qDir: %s, rDir: %s",
            //          ret.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return ret;
        }
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MicrofacetBSDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        MicrofacetBSDF &p = *(MicrofacetBSDF*)params;

        bool entering = query.dirLocal.z >= 0.0f;

        const RGBSpectrum &eEnter = entering ? p.etaExt : p.etaInt;
        const RGBSpectrum &eExit = entering ? p.etaInt : p.etaExt;
        FresnelDielectric fresnel(eEnter, eExit);

        float alphaX = p.roughnessX * p.roughnessX;
        float alphaY = p.roughnessY * p.roughnessY;
        GGXMicrofacetDistribution ggx(alphaX, alphaY);

        Vector3D dirL = entering ? dirLocal : -dirLocal;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
        float dotNVdotNL = dirL.z * dirV.z;

        if (dotNVdotNL > 0 && query.dirTypeFilter.matches(DirectionType::Reflection() | DirectionType::AllFreq())) {
            Normal3D m = halfVector(dirV, dirL);
            float dotHV = dot(dirV, m);
            float D = ggx.evaluate(m);

            RGBSpectrum F = fresnel.evaluate(dotHV);
            float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
            RGBSpectrum fs = F * D * G / (4 * dotNVdotNL);

            //VLRAssert(fs.allFinite(), "fs: %s, F: %s, G, %g, D: %g, wlIdx: %u, qDir: %s, dir: %s",
            //          fs.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return fs;
        }
        else if (dotNVdotNL < 0 && query.dirTypeFilter.matches(DirectionType::Transmission() | DirectionType::AllFreq())) {
            RGBSpectrum ret = RGBSpectrum::Zero();
            for (int wlIdx = 0; wlIdx < RGBSpectrum::NumComponents(); ++wlIdx) {
                Normal3D m_wl = normalize(-(eEnter[wlIdx] * dirV + eExit[wlIdx] * dirL) * (entering ? 1 : -1));
                float dotHV_wl = dot(dirV, m_wl);
                float dotHL_wl = dot(dirL, m_wl);
                float F_wl = fresnel.evaluate(dotHV_wl, wlIdx);
                float G_wl = ggx.evaluateSmithG1(dirV, m_wl) * ggx.evaluateSmithG1(dirL, m_wl);
                float D_wl = ggx.evaluate(m_wl);
                ret[wlIdx] = std::fabs(dotHV_wl * dotHL_wl) * (1 - F_wl) * G_wl * D_wl / std::pow(eEnter[wlIdx] * dotHV_wl + eExit[wlIdx] * dotHL_wl, 2);

                //VLRAssert(std::isfinite(ret[wlIdx]), "fs: %g, F: %g, G, %g, D: %g, wlIdx: %u, qDir: %s, dir: %s",
                //          ret[wlIdx], F_wl, G_wl, D_wl, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());
            }
            ret /= std::fabs(dotNVdotNL);
            ret *= eEnter * eEnter;
            //ret *= query.adjoint ? (eExit * eExit) : (eEnter * eEnter);// !adjoint: eExit^2 * (eEnter / eExit)^2

            //VLRAssert(ret.allFinite(), "fs: %s, wlIdx: %u, qDir: %s, dir: %s",
            //          ret.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return ret;
        }

        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float MicrofacetBSDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        MicrofacetBSDF &p = *(MicrofacetBSDF*)params;

        bool entering = query.dirLocal.z >= 0.0f;

        const RGBSpectrum &eEnter = entering ? p.etaExt : p.etaInt;
        const RGBSpectrum &eExit = entering ? p.etaInt : p.etaExt;
        FresnelDielectric fresnel(eEnter, eExit);

        float alphaX = p.roughnessX * p.roughnessX;
        float alphaY = p.roughnessY * p.roughnessY;
        GGXMicrofacetDistribution ggx(alphaX, alphaY);

        Vector3D dirL = entering ? dirLocal : -dirLocal;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
        float dotNVdotNL = dirL.z * dirV.z;
        if (dotNVdotNL == 0)
            return 0.0f;

        Normal3D m;
        if (dotNVdotNL > 0)
            m = halfVector(dirV, dirL);
        else
            m = normalize(-(eEnter[query.wlHint] * dirV + eExit[query.wlHint] * dirL));
        float dotHV = dot(dirV, m);
        if (dotHV <= 0)
            return 0.0f;
        float mPDF = ggx.evaluatePDF(dirV, m);

        RGBSpectrum F = fresnel.evaluate(dotHV);
        float reflectProb = F.importance(query.wlHint);
        if (query.dirTypeFilter.isReflection())
            reflectProb = 1.0f;
        if (query.dirTypeFilter.isTransmission())
            reflectProb = 0.0f;
        if (dotNVdotNL > 0) {
            float commonPDFTerm = reflectProb / (4 * dotHV);

            //VLRAssert(std::isfinite(commonPDFTerm) && std::isfinite(mPDF),
            //          "commonPDFTerm: %g, mPDF: %g, F: %s, wlIdx: %u, qDir: %s, dir: %s",
            //          commonPDFTerm, mPDF, F.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return commonPDFTerm * mPDF;
        }
        else {
            float dotHL = dot(dirL, m);
            float commonPDFTerm = (1 - reflectProb) / std::pow(eEnter[query.wlHint] * dotHV + eExit[query.wlHint] * dotHL, 2);

            //VLRAssert(std::isfinite(commonPDFTerm) && std::isfinite(mPDF),
            //          "commonPDFTerm: %g, mPDF: %g, F: %s, wlIdx: %u, qDir: %s, dir: %s",
            //          commonPDFTerm, mPDF, F.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return commonPDFTerm * mPDF * eExit[query.wlHint] * eExit[query.wlHint] * std::fabs(dotHL);
        }
    }

    RT_CALLABLE_PROGRAM float MicrofacetBSDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        MicrofacetBSDF &p = *(MicrofacetBSDF*)params;
        return p.coeff.importance(query.wlHint);
    }

    // END: MicrofacetBSDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // UE4 (Modified) BRDF

    struct UE4BRDF {
        RGBSpectrum baseColor;
        float roughness;
        float metallic;
    };

    RT_CALLABLE_PROGRAM uint32_t UE4SurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        UE4BRDF &p = *(UE4BRDF*)params;
        const UE4SurfaceMaterial &mat = *(const UE4SurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        Point3D texCoord = textureMap(mat.texMap, surfPt);

        optix::float4 texValue;
        texValue = optix::rtTex2D<optix::float4>(mat.texBaseColor, texCoord.x, texCoord.y);
        p.baseColor = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.texOcclusionRoughnessMetallic, texCoord.x, texCoord.y);
        p.roughness = texValue.y;
        p.metallic = texValue.z;

        return sizeof(UE4BRDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum UE4BRDF_getBaseColor(const uint32_t* params) {
        UE4BRDF &p = *(UE4BRDF*)params;

        return p.baseColor;
    }

    RT_CALLABLE_PROGRAM bool UE4BRDF_matches(const uint32_t* params, DirectionType flags) {
        DirectionType m_type = DirectionType::Reflection() | DirectionType::LowFreq() | DirectionType::HighFreq();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum UE4BRDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        UE4BRDF &p = *(UE4BRDF*)params;

        const float specular = 0.5f;
        float alpha = p.roughness * p.roughness;
        GGXMicrofacetDistribution ggx(alpha, alpha);

        bool entering = query.dirLocal.z >= 0.0f;
        Vector3D dirL;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

        float expectedF_D90 = 0.5f * p.roughness + 2 * p.roughness * query.dirLocal.z * query.dirLocal.z;
        float oneMinusDotVN5 = std::pow(1 - dirV.z, 5);
        float expectedDiffuseFresnel = lerp(1.0f, expectedF_D90, oneMinusDotVN5);
        float iBaseColor = p.baseColor.importance(query.wlHint) * expectedDiffuseFresnel * expectedDiffuseFresnel * lerp(1.0f, 1.0f / 1.51f, p.roughness);

        RGBSpectrum specularF0Color = lerp(0.08f * specular * RGBSpectrum::One(), p.baseColor, p.metallic);
        float expectedOneMinusDotVH5 = std::pow(1 - dirV.z, 5);
        float iSpecularF0 = specularF0Color.importance(query.wlHint);

        float diffuseWeight = iBaseColor * (1 - p.metallic);
        float specularWeight = lerp(iSpecularF0, 1.0f, expectedOneMinusDotVH5);

        float weights[] = { diffuseWeight, specularWeight };
        float probSelection;
        float sumWeights = 0.0f;
        uint32_t component = sampleDiscrete(weights, 2, uComponent, &probSelection, &sumWeights, &uComponent);

        float diffuseDirPDF, specularDirPDF;
        RGBSpectrum fs;
        Normal3D m;
        float dotLH;
        float D;
        if (component == 0) {
            result->sampledType = DirectionType::Reflection() | DirectionType::LowFreq();

            // JP: コサイン分布からサンプルする。
            // EN: sample based on cosine distribution.
            dirL = cosineSampleHemisphere(uDir[0], uDir[1]);
            diffuseDirPDF = dirL.z / M_PIf;

            // JP: 同じ方向サンプルを別の要素からサンプルする確率密度を求める。
            // EN: calculate PDFs to generate the sampled direction from the other distributions.
            m = halfVector(dirL, dirV);
            dotLH = dot(dirL, m);
            float commonPDFTerm = 1.0f / (4 * dotLH);
            specularDirPDF = commonPDFTerm * ggx.evaluatePDF(dirV, m);

            D = ggx.evaluate(m);
        }
        else if (component == 1) {
            result->sampledType = DirectionType::Reflection() | DirectionType::HighFreq();

            // ----------------------------------------------------------------
            // JP: ベーススペキュラー層のマイクロファセット分布からサンプルする。
            // EN: sample based on the base specular microfacet distribution.
            float mPDF;
            D = ggx.sample(dirV, uDir[0], uDir[1], &m, &mPDF);
            float dotVH = dot(dirV, m);
            dotLH = dotVH;
            dirL = 2 * dotVH * m - dirV;
            if (dirL.z * dirV.z <= 0) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            float commonPDFTerm = 1.0f / (4 * dotLH);
            specularDirPDF = commonPDFTerm * mPDF;
            // ----------------------------------------------------------------

            // JP: 同じ方向サンプルを別の要素からサンプルする確率密度を求める。
            // EN: calculate PDFs to generate the sampled direction from the other distributions.
            diffuseDirPDF = dirL.z / M_PIf;
        }

        float oneMinusDotLH5 = std::pow(1 - dotLH, 5);

        float G = ggx.evaluateSmithG1(dirL, m) * ggx.evaluateSmithG1(dirV, m);
        RGBSpectrum F = lerp(specularF0Color, RGBSpectrum::One(), oneMinusDotLH5);

        float microfacetDenom = 4 * dirL.z * dirV.z;
        RGBSpectrum specularValue = F * ((D * G) / microfacetDenom);

        float F_D90 = 0.5f * p.roughness + 2 * p.roughness * dotLH * dotLH;
        float oneMinusDotLN5 = std::pow(1 - dirL.z, 5);
        float diffuseFresnelOut = lerp(1.0f, F_D90, oneMinusDotVN5);
        float diffuseFresnelIn = lerp(1.0f, F_D90, oneMinusDotLN5);
        RGBSpectrum diffuseValue = p.baseColor * ((diffuseFresnelOut * diffuseFresnelIn * lerp(1.0f, 1.0f / 1.51f, p.roughness) / M_PIf) * (1 - p.metallic));

        RGBSpectrum ret = diffuseValue + specularValue;

        result->dirLocal = entering ? dirL : -dirL;

        // PDF based on the single-sample model MIS.
        result->dirPDF = (diffuseDirPDF * diffuseWeight + specularDirPDF * specularWeight) / sumWeights;

        return ret;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum UE4BRDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        UE4BRDF &p = *(UE4BRDF*)params;

        const float specular = 0.5f;
        float alpha = p.roughness * p.roughness;
        GGXMicrofacetDistribution ggx(alpha, alpha);

        if (dirLocal.z * query.dirLocal.z <= 0) {
            return RGBSpectrum::Zero();
        }

        bool entering = query.dirLocal.z >= 0.0f;
        Vector3D dirL = entering ? dirLocal : -dirLocal;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

        Normal3D m = halfVector(dirL, dirV);
        float dotLH = dot(dirL, m);

        float oneMinusDotLH5 = std::pow(1 - dotLH, 5);

        RGBSpectrum specularF0Color = lerp(0.08f * specular * RGBSpectrum::One(), p.baseColor, p.metallic);

        float D = ggx.evaluate(m);
        float G = ggx.evaluateSmithG1(dirL, m) * ggx.evaluateSmithG1(dirV, m);
        RGBSpectrum F = lerp(specularF0Color, RGBSpectrum::One(), oneMinusDotLH5);

        float microfacetDenom = 4 * dirL.z * dirV.z;
        RGBSpectrum specularValue = F * ((D * G) / microfacetDenom);

        float F_D90 = 0.5f * p.roughness + 2 * p.roughness * dotLH * dotLH;
        float oneMinusDotVN5 = std::pow(1 - dirV.z, 5);
        float oneMinusDotLN5 = std::pow(1 - dirL.z, 5);
        float diffuseFresnelOut = lerp(1.0f, F_D90, oneMinusDotVN5);
        float diffuseFresnelIn = lerp(1.0f, F_D90, oneMinusDotLN5);

        RGBSpectrum diffuseValue = p.baseColor * ((diffuseFresnelOut * diffuseFresnelIn * lerp(1.0f, 1.0f / 1.51f, p.roughness) / M_PIf) * (1 - p.metallic));

        RGBSpectrum ret = diffuseValue + specularValue;

        return ret;
    }

    RT_CALLABLE_PROGRAM float UE4BRDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        UE4BRDF &p = *(UE4BRDF*)params;

        const float specular = 0.5f;
        float alpha = p.roughness * p.roughness;
        GGXMicrofacetDistribution ggx(alpha, alpha);

        bool entering = query.dirLocal.z >= 0.0f;
        Vector3D dirL = entering ? dirLocal : -dirLocal;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

        Normal3D m = halfVector(dirL, dirV);
        float dotLH = dot(dirL, m);
        float commonPDFTerm = 1.0f / (4 * dotLH);

        float expectedF_D90 = 0.5f * p.roughness + 2 * p.roughness * query.dirLocal.z * query.dirLocal.z;
        float oneMinusDotVN5 = std::pow(1 - dirV.z, 5);
        float expectedDiffuseFresnel = lerp(1.0f, expectedF_D90, oneMinusDotVN5);
        float iBaseColor = p.baseColor.importance(query.wlHint) * expectedDiffuseFresnel * expectedDiffuseFresnel * lerp(1.0f, 1.0f / 1.51f, p.roughness);

        RGBSpectrum specularF0Color = lerp(0.08f * specular * RGBSpectrum::One(), p.baseColor, p.metallic);
        float expectedOneMinusDotVH5 = std::pow(1 - dirV.z, 5);
        float iSpecularF0 = specularF0Color.importance(query.wlHint);

        float diffuseWeight = iBaseColor * (1 - p.metallic);
        float specularWeight = lerp(iSpecularF0, 1.0f, expectedOneMinusDotVH5);

        float sumWeights = diffuseWeight + specularWeight;

        float diffuseDirPDF = dirL.z / M_PIf;
        float specularDirPDF = commonPDFTerm * ggx.evaluatePDF(dirV, m);

        float ret = (diffuseDirPDF * diffuseWeight + specularDirPDF * specularWeight) / sumWeights;

        return ret;
    }

    RT_CALLABLE_PROGRAM float UE4BRDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        UE4BRDF &p = *(UE4BRDF*)params;

        const float specular = 0.5f;

        bool entering = query.dirLocal.z >= 0.0f;
        Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

        float expectedF_D90 = 0.5f * p.roughness + 2 * p.roughness * query.dirLocal.z * query.dirLocal.z;
        float oneMinusDotVN5 = std::pow(1 - dirV.z, 5);
        float expectedDiffuseFresnel = lerp(1.0f, expectedF_D90, oneMinusDotVN5);
        float iBaseColor = p.baseColor.importance(query.wlHint) * expectedDiffuseFresnel * expectedDiffuseFresnel * lerp(1.0f, 1.0f / 1.51f, p.roughness);

        RGBSpectrum specularF0Color = lerp(0.08f * specular * RGBSpectrum::One(), p.baseColor, p.metallic);
        float expectedOneMinusDotVH5 = std::pow(1 - dirV.z, 5);
        float iSpecularF0 = specularF0Color.importance(query.wlHint);

        float diffuseWeight = iBaseColor * (1 - p.metallic);
        float specularWeight = lerp(iSpecularF0, 1.0f, expectedOneMinusDotVH5);

        return diffuseWeight + specularWeight;
    }

    // END: UE4 (Modified) BRDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // NullEDF

    RT_CALLABLE_PROGRAM uint32_t NullEDF_setupEDF(const uint32_t* matDesc, const SurfacePoint &surfPt, uint32_t* params) {
        return 0;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullEDF_evaluateEmittanceInternal(const uint32_t* params) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullEDF_evaluateEDFInternal(const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    // END: NullEDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // DiffuseEDF

    struct DiffuseEDF {
        RGBSpectrum emittance;
    };

    RT_CALLABLE_PROGRAM uint32_t DiffuseEmitterSurfaceMaterial_setupEDF(const uint32_t* matDesc, const SurfacePoint &surfPt, uint32_t* params) {
        DiffuseEDF &p = *(DiffuseEDF*)params;
        const DiffuseEmitterSurfaceMaterial &mat = *(const DiffuseEmitterSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        Point3D texCoord = textureMap(mat.texMap, surfPt);

        optix::float4 texValue = optix::rtTex2D<optix::float4>(mat.texEmittance, texCoord.x, texCoord.y);
        p.emittance = RGBSpectrum(texValue.x, texValue.y, texValue.z);

        return sizeof(DiffuseEDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum DiffuseEDF_evaluateEmittanceInternal(const uint32_t* params) {
        DiffuseEDF &p = *(DiffuseEDF*)params;
        return p.emittance;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum DiffuseEDF_evaluateEDFInternal(const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum(dirLocal.z > 0.0f ? 1.0f / M_PIf : 0.0f);
    }

    // END: DiffuseEDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // MultiBSDF / MultiEDF

    // bsdf0-3: param offsets
    // numBSDFs
    // --------------------------------
    // BSDF0 procedure set index
    // BSDF0 params
    // ...
    // BSDF3 procedure set index
    // BSDF3 params
    struct MultiBSDF {
        struct {
            unsigned int bsdf0 : 6;
            unsigned int bsdf1 : 6;
            unsigned int bsdf2 : 6;
            unsigned int bsdf3 : 6;
            unsigned int numBSDFs : 8;
        };
    };

    RT_CALLABLE_PROGRAM uint32_t MultiSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        MultiBSDF &p = *(MultiBSDF*)params;
        const MultiSurfaceMaterial &mat = *(const MultiSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        uint32_t baseIndex = sizeof(MultiBSDF) / 4;
        const uint32_t matOffsets[] = { mat.matOffset0, mat.matOffset1, mat.matOffset2, mat.matOffset3 };
        uint32_t bsdfOffsets[4] = { 0, 0, 0, 0 };
        for (int i = 0; i < mat.numMaterials; ++i) {
            bsdfOffsets[i] = baseIndex;

            const SurfaceMaterialHead &matHead = *(const SurfaceMaterialHead*)(matDesc + matOffsets[i]);
            //rtPrintf("%d: %u, %u, %u, %u\n", i, matHead.progSetupBSDF, matHead.bsdfProcedureSetIndex, matHead.progSetupEDF, matHead.edfProcedureSetIndex);
            progSigSetupBSDF setupBSDF = (progSigSetupBSDF)matHead.progSetupBSDF;
            *(uint32_t*)(params + baseIndex++) = matHead.bsdfProcedureSetIndex;
            baseIndex += setupBSDF((const uint32_t*)&matHead, surfPt, wavelengthSelected, params + baseIndex);
        }

        p.bsdf0 = bsdfOffsets[0];
        p.bsdf1 = bsdfOffsets[1];
        p.bsdf2 = bsdfOffsets[2];
        p.bsdf3 = bsdfOffsets[3];
        p.numBSDFs = mat.numMaterials;

        //rtPrintf("%u, %u, %u, %u, %u mats\n", p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3, p.numBSDFs);

        return baseIndex;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiBSDF_getBaseColor(const uint32_t* params) {
        const MultiBSDF &p = *(const MultiBSDF*)params;

        uint32_t bsdfOffsets[4] = { p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3 };

        RGBSpectrum ret;
        for (int i = 0; i < p.numBSDFs; ++i) {
            const uint32_t* bsdf = params + bsdfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)bsdf;
            const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
            progSigGetBaseColor getBaseColor = (progSigGetBaseColor)procSet.progGetBaseColor;

            ret += getBaseColor(bsdf + 1);
        }

        return ret;
    }

    RT_CALLABLE_PROGRAM bool MultiBSDF_matches(const uint32_t* params, DirectionType flags) {
        const MultiBSDF &p = *(const MultiBSDF*)params;

        uint32_t bsdfOffsets[4] = { p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3 };

        for (int i = 0; i < p.numBSDFs; ++i) {
            const uint32_t* bsdf = params + bsdfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)bsdf;
            const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
            progSigBSDFmatches matches = (progSigBSDFmatches)procSet.progBSDFmatches;

            if (matches(bsdf + 1, flags))
                return true;
        }

        return false;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiBSDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        const MultiBSDF &p = *(const MultiBSDF*)params;

        uint32_t bsdfOffsets[4] = { p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3 };

        float weights[4];
        for (int i = 0; i < p.numBSDFs; ++i) {
            const uint32_t* bsdf = params + bsdfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)bsdf;
            const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
            progSigBSDFWeightInternal weightInternal = (progSigBSDFWeightInternal)procSet.progWeightInternal;

            weights[i] = weightInternal(bsdf + 1, query);
        }

        // JP: 各BSDFのウェイトに基づいて方向のサンプルを行うBSDFを選択する。
        // EN: Based on the weight of each BSDF, select a BSDF from which direction sampling.
        float tempProb;
        float sumWeights;
        uint32_t idx = sampleDiscrete(weights, p.numBSDFs, uComponent, &tempProb, &sumWeights, &uComponent);
        if (sumWeights == 0.0f) {
            result->dirPDF = 0.0f;
            return RGBSpectrum::Zero();
        }

        const uint32_t* selectedBSDF = params + bsdfOffsets[idx];
        uint32_t selProcIdx = *(const uint32_t*)selectedBSDF;
        const BSDFProcedureSet selProcSet = pv_bsdfProcedureSetBuffer[selProcIdx];
        progSigSampleBSDFInternal sampleInternal = (progSigSampleBSDFInternal)selProcSet.progSampleBSDFInternal;

        // JP: 選択したBSDFから方向をサンプリングする。
        // EN: sample a direction from the selected BSDF.
        RGBSpectrum value = sampleInternal(selectedBSDF + 1, query, uComponent, uDir, result);
        result->dirPDF *= weights[idx];
        if (result->dirPDF == 0.0f) {
            result->dirPDF = 0.0f;
            return RGBSpectrum::Zero();
        }

        // JP: サンプルした方向に関するBSDFの値の合計と、single-sample model MISに基づいた確率密度を計算する。
        // EN: calculate the total of BSDF values and a PDF based on the single-sample model MIS for the sampled direction.
        if (!result->sampledType.isDelta()) {
            for (int i = 0; i < p.numBSDFs; ++i) {
                const uint32_t* bsdf = params + bsdfOffsets[i];
                uint32_t procIdx = *(const uint32_t*)bsdf;
                const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
                progSigBSDFmatches matches = (progSigBSDFmatches)procSet.progBSDFmatches;
                progSigEvaluateBSDF_PDFInternal evaluatePDFInternal = (progSigEvaluateBSDF_PDFInternal)procSet.progEvaluateBSDF_PDFInternal;

                if (i != idx && matches(bsdf + 1, query.dirTypeFilter))
                    result->dirPDF += evaluatePDFInternal(bsdf + 1, query, result->dirLocal) * weights[i];
            }

            BSDFQuery mQuery = query;
            mQuery.dirTypeFilter &= sideTest(query.geometricNormalLocal, query.dirLocal, result->dirLocal);
            value = RGBSpectrum::Zero();
            for (int i = 0; i < p.numBSDFs; ++i) {
                const uint32_t* bsdf = params + bsdfOffsets[i];
                uint32_t procIdx = *(const uint32_t*)bsdf;
                const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
                progSigBSDFmatches matches = (progSigBSDFmatches)procSet.progBSDFmatches;
                progSigEvaluateBSDFInternal evaluateBSDFInternal = (progSigEvaluateBSDFInternal)procSet.progEvaluateBSDFInternal;

                if (!matches(bsdf + 1, mQuery.dirTypeFilter))
                    continue;
                value += evaluateBSDFInternal(bsdf + 1, mQuery, result->dirLocal);
            }
        }
        result->dirPDF /= sumWeights;

        return value;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiBSDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        const MultiBSDF &p = *(const MultiBSDF*)params;

        uint32_t bsdfOffsets[4] = { p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3 };

        RGBSpectrum retValue = RGBSpectrum::Zero();
        for (int i = 0; i < p.numBSDFs; ++i) {
            const uint32_t* bsdf = params + bsdfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)bsdf;
            const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
            progSigBSDFmatches matches = (progSigBSDFmatches)procSet.progBSDFmatches;
            progSigEvaluateBSDFInternal evaluateBSDFInternal = (progSigEvaluateBSDFInternal)procSet.progEvaluateBSDFInternal;

            if (!matches(bsdf + 1, query.dirTypeFilter))
                continue;
            retValue += evaluateBSDFInternal(bsdf + 1, query, dirLocal);
        }
        return retValue;
    }

    RT_CALLABLE_PROGRAM float MultiBSDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        const MultiBSDF &p = *(const MultiBSDF*)params;

        uint32_t bsdfOffsets[4] = { p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3 };

        float sumWeights = 0.0f;
        float weights[4];
        for (int i = 0; i < p.numBSDFs; ++i) {
            const uint32_t* bsdf = params + bsdfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)bsdf;
            const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
            progSigBSDFWeightInternal weightInternal = (progSigBSDFWeightInternal)procSet.progWeightInternal;

            weights[i] = weightInternal(bsdf + 1, query);
            sumWeights += weights[i];
        }
        if (sumWeights == 0.0f)
            return 0.0f;

        float retPDF = 0.0f;
        for (int i = 0; i < p.numBSDFs; ++i) {
            const uint32_t* bsdf = params + bsdfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)bsdf;
            const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
            progSigEvaluateBSDF_PDFInternal evaluatePDFInternal = (progSigEvaluateBSDF_PDFInternal)procSet.progEvaluateBSDF_PDFInternal;

            if (weights[i] > 0)
                retPDF += evaluatePDFInternal(bsdf + 1, query, dirLocal) * weights[i];
        }
        retPDF /= sumWeights;

        return retPDF;
    }

    RT_CALLABLE_PROGRAM float MultiBSDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        const MultiBSDF &p = *(const MultiBSDF*)params;

        uint32_t bsdfOffsets[4] = { p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3 };

        float ret = 0.0f;
        for (int i = 0; i < p.numBSDFs; ++i) {
            const uint32_t* bsdf = params + bsdfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)bsdf;
            const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[procIdx];
            progSigBSDFWeightInternal weightInternal = (progSigBSDFWeightInternal)procSet.progWeightInternal;

            ret += weightInternal(bsdf + 1, query);
        }

        return ret;
    }

    // edf0-3: param offsets
    // numEDFs
    // --------------------------------
    // EDF0 procedure set index
    // EDF0 params
    // ...
    // EDF3 procedure set index
    // EDF3 params
    struct MultiEDF {
        struct {
            unsigned int edf0 : 6;
            unsigned int edf1 : 6;
            unsigned int edf2 : 6;
            unsigned int edf3 : 6;
            unsigned int numEDFs : 8;
        };
    };

    RT_CALLABLE_PROGRAM uint32_t MultiSurfaceMaterial_setupEDF(const uint32_t* matDesc, const SurfacePoint &surfPt, uint32_t* params) {
        MultiEDF &p = *(MultiEDF*)params;
        const MultiSurfaceMaterial &mat = *(const MultiSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        uint32_t baseIndex = sizeof(MultiEDF) / 4;
        const uint32_t matOffsets[] = { mat.matOffset0, mat.matOffset1, mat.matOffset2, mat.matOffset3 };
        uint32_t edfOffsets[4] = { 0, 0, 0, 0 };
        for (int i = 0; i < mat.numMaterials; ++i) {
            edfOffsets[i] = baseIndex;

            const SurfaceMaterialHead &matHead = *(const SurfaceMaterialHead*)(matDesc + matOffsets[i]);
            progSigSetupEDF setupEDF = (progSigSetupEDF)matHead.progSetupEDF;
            *(uint32_t*)(params + baseIndex++) = matHead.edfProcedureSetIndex;
            baseIndex += setupEDF((const uint32_t*)&matHead, surfPt, params + baseIndex);
        }

        p.edf0 = edfOffsets[0];
        p.edf1 = edfOffsets[1];
        p.edf2 = edfOffsets[2];
        p.edf3 = edfOffsets[3];
        p.numEDFs = mat.numMaterials;

        return baseIndex;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiEDF_evaluateEmittanceInternal(const uint32_t* params) {
        const MultiEDF &p = *(const MultiEDF*)params;

        uint32_t edfOffsets[4] = { p.edf0, p.edf1, p.edf2, p.edf3 };

        RGBSpectrum ret = RGBSpectrum::Zero();
        for (int i = 0; i < p.numEDFs; ++i) {
            const uint32_t* edf = params + edfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)edf;
            const EDFProcedureSet procSet = pv_edfProcedureSetBuffer[procIdx];
            progSigEvaluateEmittanceInternal evaluateEmittanceInternal = (progSigEvaluateEmittanceInternal)procSet.progEvaluateEmittanceInternal;

            ret += evaluateEmittanceInternal(edf + 1);
        }

        return ret;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiEDF_evaluateEDFInternal(const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {
        const MultiEDF &p = *(const MultiEDF*)params;

        uint32_t edfOffsets[4] = { p.edf0, p.edf1, p.edf2, p.edf3 };

        RGBSpectrum ret = RGBSpectrum::Zero();
        RGBSpectrum sumEmittance = RGBSpectrum::Zero();
        for (int i = 0; i < p.numEDFs; ++i) {
            const uint32_t* edf = params + edfOffsets[i];
            uint32_t procIdx = *(const uint32_t*)edf;
            const EDFProcedureSet procSet = pv_edfProcedureSetBuffer[procIdx];
            progSigEvaluateEmittanceInternal evaluateEmittanceInternal = (progSigEvaluateEmittanceInternal)procSet.progEvaluateEmittanceInternal;
            progSigEvaluateEDFInternal evaluateEDFInternal = (progSigEvaluateEDFInternal)procSet.progEvaluateEDFInternal;

            RGBSpectrum emittance = evaluateEmittanceInternal(edf + 1);
            sumEmittance += emittance;
            ret += emittance * evaluateEDFInternal(edf + 1, query, dirLocal);
        }
        ret.safeDivide(sumEmittance);

        return ret;
    }

    // END: MultiBSDF / MultiEDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // EnvironmentEDF

    struct EnvironmentEDF {
        RGBSpectrum emittance;
    };

    RT_CALLABLE_PROGRAM uint32_t EnvironmentEmitterSurfaceMaterial_setupEDF(const uint32_t* matDesc, const SurfacePoint &surfPt, uint32_t* params) {
        EnvironmentEDF &p = *(EnvironmentEDF*)params;
        const EnvironmentEmitterSurfaceMaterial &mat = *(const EnvironmentEmitterSurfaceMaterial*)(matDesc + sizeof(SurfaceMaterialHead) / 4);

        optix::float4 texValue = optix::rtTex2D<optix::float4>(mat.texEmittance, surfPt.texCoord.u, surfPt.texCoord.v);
        p.emittance = RGBSpectrum(texValue.x, texValue.y, texValue.z);

        return sizeof(EnvironmentEDF) / 4;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum EnvironmentEDF_evaluateEmittanceInternal(const uint32_t* params) {
        EnvironmentEDF &p = *(EnvironmentEDF*)params;
        return M_PIf * p.emittance;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum EnvironmentEDF_evaluateEDFInternal(const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum(dirLocal.z > 0.0f ? 1.0f / M_PIf : 0.0f);
    }

    // END: EnvironmentEDF
    // ----------------------------------------------------------------
}
