#include "hip/hip_runtime.h"
#include "kernel_common.cuh"
#include "random_distributions.cuh"

namespace VLR {
    // ----------------------------------------------------------------
    // NullBSDF

    RT_CALLABLE_PROGRAM void NullBSDF_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullBSDF_getBaseColor(const uint32_t* params) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM bool NullBSDF_matches(DirectionType flags) {
        return false;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullBSDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullBSDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float NullBSDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float NullBSDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        return 0.0f;
    }

    // END: NullBSDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // MatteBRDF

    struct MatteBRDF {
        RGBSpectrum albedo;
        float roughness;
    };

    RT_CALLABLE_PROGRAM void MatteSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        MatteBRDF &p = *(MatteBRDF*)params;
        const MatteSurfaceMaterial &mat = *(const MatteSurfaceMaterial*)matDesc;

        optix::float4 texValue = optix::rtTex2D<optix::float4>(mat.texAlbedoRoughness, surfPt.texCoord.u, surfPt.texCoord.v);
        p.albedo = sRGB_degamma(RGBSpectrum(texValue.x, texValue.y, texValue.z));
        p.roughness = texValue.w;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MatteBRDF_getBaseColor(const uint32_t* params) {
        MatteBRDF &p = *(MatteBRDF*)params;

        return p.albedo;
    }

    RT_CALLABLE_PROGRAM bool MatteBRDF_matches(DirectionType flags) {
        DirectionType m_type = DirectionType::Reflection() | DirectionType::LowFreq();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MatteBRDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        MatteBRDF &p = *(MatteBRDF*)params;

        result->dirLocal = cosineSampleHemisphere(uDir[0], uDir[1]);
        result->dirPDF = result->dirLocal.z / M_PIf;
        result->sampledType = DirectionType::Reflection() | DirectionType::LowFreq();
        result->dirLocal.z *= query.dirLocal.z > 0 ? 1 : -1;

        return p.albedo / M_PIf;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MatteBRDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        MatteBRDF &p = *(MatteBRDF*)params;

        if (query.dirLocal.z * dirLocal.z <= 0.0f) {
            RGBSpectrum fs = RGBSpectrum::Zero();
            return fs;
        }
        RGBSpectrum fs = p.albedo / M_PIf;

        return fs;
    }

    RT_CALLABLE_PROGRAM float MatteBRDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        if (query.dirLocal.z * dirLocal.z <= 0.0f) {
            return 0.0f;
        }

        return std::abs(dirLocal.z) / M_PIf;
    }

    RT_CALLABLE_PROGRAM float MatteBRDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        MatteBRDF &p = *(MatteBRDF*)params;
        return p.albedo.importance(query.wlHint);
    }

    // END: MatteBRDF
    // ----------------------------------------------------------------



    RT_FUNCTION RGBSpectrum evaluateFresnelConductor(const RGBSpectrum &eta, const RGBSpectrum &k, float cosEnter) {
        cosEnter = std::fabs(cosEnter);
        float cosEnter2 = cosEnter * cosEnter;
        RGBSpectrum _2EtaCosEnter = 2.0f * eta * cosEnter;
        RGBSpectrum tmp_f = eta * eta + k * k;
        RGBSpectrum tmp = tmp_f * cosEnter2;
        RGBSpectrum Rparl2 = (tmp - _2EtaCosEnter + 1) / (tmp + _2EtaCosEnter + 1);
        RGBSpectrum Rperp2 = (tmp_f - _2EtaCosEnter + cosEnter2) / (tmp_f + _2EtaCosEnter + cosEnter2);
        return (Rparl2 + Rperp2) / 2.0f;
    }

    RT_FUNCTION float evalF(float etaEnter, float etaExit, float cosEnter, float cosExit) {
        float Rparl = ((etaExit * cosEnter) - (etaEnter * cosExit)) / ((etaExit * cosEnter) + (etaEnter * cosExit));
        float Rperp = ((etaEnter * cosEnter) - (etaExit * cosExit)) / ((etaEnter * cosEnter) + (etaExit * cosExit));
        return (Rparl * Rparl + Rperp * Rperp) / 2.0f;
    }

    RT_FUNCTION RGBSpectrum evaluateFresnelDielectric(const RGBSpectrum &etaExt, const RGBSpectrum &etaInt, float cosEnter) {
        cosEnter = std::fmin(std::fmax(cosEnter, -1.0f), 1.0f);

        bool entering = cosEnter > 0.0f;
        const RGBSpectrum &eEnter = entering ? etaExt : etaInt;
        const RGBSpectrum &eExit = entering ? etaInt : etaExt;

        RGBSpectrum sinExit = eEnter / eExit * std::sqrt(std::fmax(0.0f, 1.0f - cosEnter * cosEnter));
        RGBSpectrum ret = RGBSpectrum::Zero();
        cosEnter = std::fabs(cosEnter);
        for (int i = 0; i < RGBSpectrum::NumComponents(); ++i) {
            if (sinExit[i] >= 1.0f) {
                ret[i] = 1.0f;
            }
            else {
                float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit[i] * sinExit[i]));
                ret[i] = evalF(eEnter[i], eExit[i], cosEnter, cosExit);
            }
        }

        return ret;
    }



    // ----------------------------------------------------------------
    // SpecularBRDF

    struct SpecularBRDF {
        RGBSpectrum coeffR;
        RGBSpectrum eta;
        RGBSpectrum k;
    };

    RT_CALLABLE_PROGRAM void SpecularReflectionSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        SpecularBRDF &p = *(SpecularBRDF*)params;
        const SpecularReflectionSurfaceMaterial &mat = *(const SpecularReflectionSurfaceMaterial*)matDesc;

        optix::float4 texValue;
        texValue = optix::rtTex2D<optix::float4>(mat.texCoeffR, surfPt.texCoord.u, surfPt.texCoord.v);
        p.coeffR = sRGB_degamma(RGBSpectrum(texValue.x, texValue.y, texValue.z));
        texValue = optix::rtTex2D<optix::float4>(mat.texEta, surfPt.texCoord.u, surfPt.texCoord.v);
        p.eta = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.tex_k, surfPt.texCoord.u, surfPt.texCoord.v);
        p.k = RGBSpectrum(texValue.x, texValue.y, texValue.z);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBRDF_getBaseColor(const uint32_t* params) {
        SpecularBRDF &p = *(SpecularBRDF*)params;

        return p.coeffR;
    }

    RT_CALLABLE_PROGRAM bool SpecularBRDF_matches(DirectionType flags) {
        DirectionType m_type = DirectionType::Reflection() | DirectionType::Delta0D();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBRDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        SpecularBRDF &p = *(SpecularBRDF*)params;

        result->dirLocal = Vector3D(-query.dirLocal.x, -query.dirLocal.y, query.dirLocal.z);
        result->dirPDF = 1.0f;
        result->sampledType = DirectionType::Reflection() | DirectionType::Delta0D();
        RGBSpectrum fs = p.coeffR * evaluateFresnelConductor(p.eta, p.k, query.dirLocal.z) / std::fabs(query.dirLocal.z);

        return fs;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBRDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float SpecularBRDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float SpecularBRDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        SpecularBRDF &p = *(SpecularBRDF*)params;
        float ret = (p.coeffR * evaluateFresnelConductor(p.eta, p.k, query.dirLocal.z)).importance(query.wlHint);
        //float snCorrection = query.adjoint ? std::fabs(dot(Vector3D(-query.dirLocal.x, -query.dirLocal.y, query.dirLocal.z), query.gNormalLocal) /
        //                                               query.dirLocal.z) : 1;
        return ret;
    }

    // END: SpecularBRDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // SpecularBSDF

    struct SpecularBSDF {
        RGBSpectrum coeff;
        RGBSpectrum etaExt;
        RGBSpectrum etaInt;
        bool dispersive;
    };

    RT_CALLABLE_PROGRAM void SpecularScatteringSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        SpecularBSDF &p = *(SpecularBSDF*)params;
        const SpecularScatteringSurfaceMaterial &mat = *(const SpecularScatteringSurfaceMaterial*)matDesc;

        optix::float4 texValue;
        texValue = optix::rtTex2D<optix::float4>(mat.texCoeff, surfPt.texCoord.u, surfPt.texCoord.v);
        p.coeff = sRGB_degamma(RGBSpectrum(texValue.x, texValue.y, texValue.z));
        texValue = optix::rtTex2D<optix::float4>(mat.texEtaExt, surfPt.texCoord.u, surfPt.texCoord.v);
        p.etaExt = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        texValue = optix::rtTex2D<optix::float4>(mat.texEtaInt, surfPt.texCoord.u, surfPt.texCoord.v);
        p.etaInt = RGBSpectrum(texValue.x, texValue.y, texValue.z);
        p.dispersive = !wavelengthSelected;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBSDF_getBaseColor(const uint32_t* params) {
        SpecularBSDF &p = *(SpecularBSDF*)params;

        return p.coeff;
    }

    RT_CALLABLE_PROGRAM bool SpecularBSDF_matches(DirectionType flags) {
        DirectionType m_type = DirectionType::WholeSphere() | DirectionType::Delta0D();
        return m_type.matches(flags);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBSDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        SpecularBSDF &p = *(SpecularBSDF*)params;

        RGBSpectrum F = evaluateFresnelDielectric(p.etaExt, p.etaInt, query.dirLocal.z);
        float reflectProb = F.importance(query.wlHint);
        if (query.dirTypeFilter.isReflection())
            reflectProb = 1.0f;
        if (query.dirTypeFilter.isTransmission())
            reflectProb = 0.0f;
        if (uComponent < reflectProb) {
            if (query.dirLocal.z == 0.0f) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            result->dirLocal = Vector3D(-query.dirLocal.x, -query.dirLocal.y, query.dirLocal.z);
            result->dirPDF = reflectProb;
            result->sampledType = DirectionType::Reflection() | DirectionType::Delta0D();
            RGBSpectrum fs = p.coeff * F / std::fabs(query.dirLocal.z);

            return fs;
        }
        else {
            bool entering = query.dirLocal.z > 0.0f;
            float eEnter = entering ? p.etaExt[query.wlHint] : p.etaInt[query.wlHint];
            float eExit = entering ? p.etaInt[query.wlHint] : p.etaExt[query.wlHint];

            float sinEnter2 = 1.0f - query.dirLocal.z * query.dirLocal.z;
            float rrEta = eEnter / eExit;// reciprocal of relative IOR.
            float sinExit2 = rrEta * rrEta * sinEnter2;

            if (sinExit2 >= 1.0f) {
                result->dirPDF = 0.0f;
                return RGBSpectrum::Zero();
            }
            float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit2));
            if (entering)
                cosExit = -cosExit;
            result->dirLocal = Vector3D(rrEta * -query.dirLocal.x, rrEta * -query.dirLocal.y, cosExit);
            result->dirPDF = 1.0f - reflectProb;
            result->sampledType = DirectionType::Transmission() | DirectionType::Delta0D() | (p.dispersive ? DirectionType::Dispersive() : DirectionType());
            RGBSpectrum ret = RGBSpectrum::Zero();
            ret[query.wlHint] = p.coeff[query.wlHint] * (1.0f - F[query.wlHint]);
            RGBSpectrum fs = ret / std::fabs(cosExit);

            return fs;
        }
    }

    RT_CALLABLE_PROGRAM RGBSpectrum SpecularBSDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float SpecularBSDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float SpecularBSDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        SpecularBSDF &p = *(SpecularBSDF*)params;
        return p.coeff.importance(query.wlHint);
    }

    // END: SpecularBSDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // UE4 BRDF

    struct UE4BRDF {
        RGBSpectrum baseColor;
        float roughenss;
        float metallic;
    };

    RT_CALLABLE_PROGRAM void UE4SurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        UE4BRDF &p = *(UE4BRDF*)params;
        const UE4SurfaceMaterial &mat = *(const UE4SurfaceMaterial*)matDesc;

        VLRAssert_NotImplemented();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum UE4BRDF_getBaseColor(const uint32_t* params) {
        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }
    
    RT_CALLABLE_PROGRAM bool UE4BRDF_matches(DirectionType flags) {
        VLRAssert_NotImplemented();
        return true;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum UE4BRDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum UE4BRDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float UE4BRDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        VLRAssert_NotImplemented();
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float UE4BRDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        VLRAssert_NotImplemented();
        return 0.0f;
    }

    // END: UE4 BRDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // NullEDF

    RT_CALLABLE_PROGRAM void NullEDF_setupEDF(const uint32_t* matDesc, const SurfacePoint &surfPt, uint32_t* params) {
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullEDF_evaluateEmittanceInternal(const uint32_t* params) {
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum NullEDF_evaluateEDFInternal(const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum::Zero();
    }

    // END: NullEDF
    // ----------------------------------------------------------------


    
    // ----------------------------------------------------------------
    // DiffuseEDF

    struct DiffuseEDF {
        RGBSpectrum emittance;
    };

    RT_CALLABLE_PROGRAM void DiffuseEmitterSurfaceMaterial_setupEDF(const uint32_t* matDesc, const SurfacePoint &surfPt, uint32_t* params) {
        DiffuseEDF &p = *(DiffuseEDF*)params;
        const DiffuseEmitterSurfaceMaterial &mat = *(const DiffuseEmitterSurfaceMaterial*)matDesc;

        optix::float4 texValue = optix::rtTex2D<optix::float4>(mat.texEmittance, surfPt.texCoord.u, surfPt.texCoord.v);
        p.emittance = RGBSpectrum(texValue.x, texValue.y, texValue.z);
    }

    RT_CALLABLE_PROGRAM RGBSpectrum DiffuseEDF_evaluateEmittanceInternal(const uint32_t* params) {
        DiffuseEDF &p = *(DiffuseEDF*)params;
        return p.emittance;
    }
    
    RT_CALLABLE_PROGRAM RGBSpectrum DiffuseEDF_evaluateEDFInternal(const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {
        return RGBSpectrum(dirLocal.z > 0.0f ? 1.0f / M_PIf : 0.0f);
    }

    // END: DiffuseEDF
    // ----------------------------------------------------------------



    // ----------------------------------------------------------------
    // MultiBSDF / MultiEDF

    // bsdf0-3: param offsets
    // numBSDFs
    // --------------------------------
    // BSDF0 procedure set index
    // BSDF0 params
    // ...
    // BSDF3 procedure set index
    // BSDF3 params
    struct MultiBSDF {
        struct {
            unsigned int bsdf0 : 6;
            unsigned int bsdf1 : 6;
            unsigned int bsdf2 : 6;
            unsigned int bsdf3 : 6;
            unsigned int numBSDFs : 8;
        };
    };

    RT_CALLABLE_PROGRAM void MultiSurfaceMaterial_setupBSDF(const uint32_t* matDesc, const SurfacePoint &surfPt, bool wavelengthSelected, uint32_t* params) {
        MultiBSDF &p = *(MultiBSDF*)params;
        const MultiSurfaceMaterial &mat = *(const MultiSurfaceMaterial*)matDesc;

        uint32_t baseIndex = 0;
        uint32_t bsdfOffsets[4] = { 0, 0, 0, 0 };
        for (int i = 0; i < mat.numMaterials; ++i) {

        }
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiBSDF_getBaseColor(const uint32_t* params) {
        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM bool MultiBSDF_matches(DirectionType flags) {
        VLRAssert_NotImplemented();
        return true;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiBSDF_sampleBSDFInternal(const uint32_t* params, const BSDFQuery &query, float uComponent, const float uDir[2], BSDFQueryResult* result) {
        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiBSDF_evaluateBSDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM float MultiBSDF_evaluateBSDF_PDFInternal(const uint32_t* params, const BSDFQuery &query, const Vector3D &dirLocal) {
        VLRAssert_NotImplemented();
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM float MultiBSDF_weightInternal(const uint32_t* params, const BSDFQuery &query) {
        VLRAssert_NotImplemented();
        return 0.0f;
    }

    // edf0-3: param offsets
    // numEDFs
    // --------------------------------
    // EDF0 procedure set index
    // EDF0 params
    // ...
    // EDF3 procedure set index
    // EDF3 params
    struct MultiEDF {
        struct {
            unsigned int edf0 : 6;
            unsigned int edf1 : 6;
            unsigned int edf2 : 6;
            unsigned int edf3 : 6;
            unsigned int numEDFs : 8;
        };
    };

    RT_CALLABLE_PROGRAM void MultiSurfaceMaterial_setupEDF(const uint32_t* matDesc, const SurfacePoint &surfPt, uint32_t* params) {
        MultiEDF &p = *(MultiEDF*)params;
        const MultiSurfaceMaterial &mat = *(const MultiSurfaceMaterial*)matDesc;
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiEDF_evaluateEmittanceInternal(const uint32_t* params) {
        MultiEDF &p = *(MultiEDF*)params;

        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }

    RT_CALLABLE_PROGRAM RGBSpectrum MultiEDF_evaluateEDFInternal(const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {
        VLRAssert_NotImplemented();
        return RGBSpectrum::Zero();
    }

    // END: MultiBSDF / MultiEDF
    // ----------------------------------------------------------------
}
