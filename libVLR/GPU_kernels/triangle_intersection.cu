#include "hip/hip_runtime.h"
﻿#include "kernel_common.cuh"

namespace VLR {
    // per GeometryInstance
    // closestHitProgramなどから呼ばれるdecodeHitPoint等で読み出すためにはGeometryInstanceレベルにバインドする必要がある。
    rtBuffer<Vertex> pv_vertexBuffer;
    rtBuffer<Triangle> pv_triangleBuffer;
    rtDeclareVariable(float, pv_sumImportances, , );

    // Intersection Program
    RT_PROGRAM void intersectTriangle(int32_t primIdx) {
        const Triangle &triangle = pv_triangleBuffer[primIdx];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        // use a triangle intersection function defined in optix_math_namespace.h
        optix::float3 gn;
        float t;
        float b0, b1, b2;
        if (!intersect_triangle(sm_ray, asOptiXType(v0.position), asOptiXType(v1.position), asOptiXType(v2.position),
                                gn, t, b1, b2))
            return;

        if (!rtPotentialIntersection(t))
            return;

        b0 = 1.0f - b1 - b2;
        a_hitPointParam.b0 = b0;
        a_hitPointParam.b1 = b1;
        a_hitPointParam.primIndex = primIdx;

        const uint32_t materialIndex = 0;
        rtReportIntersection(materialIndex);
    }

    // Bounding Box Program
    RT_PROGRAM void calcBBoxForTriangle(int32_t primIdx, float result[6]) {
        const Triangle &triangle = pv_triangleBuffer[primIdx];
        const Point3D &p0 = pv_vertexBuffer[triangle.index0].position;
        const Point3D &p1 = pv_vertexBuffer[triangle.index1].position;
        const Point3D &p2 = pv_vertexBuffer[triangle.index2].position;

        //optix::Aabb* bbox = (optix::Aabb*)result;
        //*bbox = optix::Aabb(asOptiXType(p0), asOptiXType(p1), asOptiXType(p2));

        BoundingBox3D* bbox = (BoundingBox3D*)result;
        *bbox = BoundingBox3D(Point3D(INFINITY), Point3D(-INFINITY));
        bbox->unify(p0);
        bbox->unify(p1);
        bbox->unify(p2);
    }



    // bound
    RT_CALLABLE_PROGRAM void decodeHitPointForTriangle(const HitPointParameter &param, SurfacePoint* surfPt, float* hypAreaPDF) {
        const Triangle &triangle = pv_triangleBuffer[param.primIndex];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        Normal3D geometricNormal = cross(v1.position - v0.position, v2.position - v0.position);
        float area = geometricNormal.length() / 2;
        geometricNormal /= 2 * area;

        geometricNormal = normalize(transform(RT_OBJECT_TO_WORLD, geometricNormal));

        // JP: プログラムがこの点を光源としてサンプルする場合の面積に関する(仮想的な)PDFを求める。
        // EN: calculate a hypothetical area PDF value in the case where the program sample this point as light.
        float probLightPrim = area / pv_sumImportances;
        *hypAreaPDF = probLightPrim / area;

        float b0 = param.b0, b1 = param.b1, b2 = 1.0f - param.b0 - param.b1;
        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        position = transform(RT_OBJECT_TO_WORLD, position);
        shadingNormal = normalize(transform(RT_OBJECT_TO_WORLD, shadingNormal));
        tc0Direction = normalize(transform(RT_OBJECT_TO_WORLD, tc0Direction));

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = normalize(tc0Direction - dotNT * shadingNormal);

        surfPt->position = position;
        surfPt->shadingFrame = ReferenceFrame(tc0Direction, shadingNormal);
        surfPt->isPoint = false;
        surfPt->atInfinity = false;
        surfPt->geometricNormal = geometricNormal;
        surfPt->u = b0;
        surfPt->v = b1;
        surfPt->texCoord = texCoord;
        surfPt->tc0Direction = tc0Direction;
    }

    // bound
    RT_CALLABLE_PROGRAM TexCoord2D decodeTexCoordForTriangle(const HitPointParameter &param) {
        const Triangle &triangle = pv_triangleBuffer[param.primIndex];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        float b0 = param.b0, b1 = param.b1, b2 = 1.0f - param.b0 - param.b1;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        return texCoord;
    }



    RT_CALLABLE_PROGRAM void sampleTriangleMesh(const SurfaceLightDescriptor::Body &desc, const SurfaceLightPosSample &sample, SurfaceLightPosQueryResult* result) {
        float primProb;
        uint32_t primIdx = desc.asMeshLight.primDistribution.sample(sample.uElem, &primProb);

        result->materialIndex = desc.asMeshLight.materialIndex;

        const Triangle &triangle = desc.asMeshLight.triangleBuffer[primIdx];
        const Vertex &v0 = desc.asMeshLight.vertexBuffer[triangle.index0];
        const Vertex &v1 = desc.asMeshLight.vertexBuffer[triangle.index1];
        const Vertex &v2 = desc.asMeshLight.vertexBuffer[triangle.index2];

        Normal3D geometricNormal = cross(v1.position - v0.position, v2.position - v0.position);
        float area = geometricNormal.length() / 2;
        geometricNormal /= 2 * area;

        geometricNormal = normalize(desc.asMeshLight.transform * geometricNormal);

        result->areaPDF = primProb / area;
        result->posType = DirectionType::Emission() | DirectionType::LowFreq();

        float b0, b1, b2;
        uniformSampleTriangle(sample.uPos[0], sample.uPos[1], &b0, &b1);
        b2 = 1.0f - b0 - b1;

        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        position = desc.asMeshLight.transform * position;
        shadingNormal = normalize(desc.asMeshLight.transform * shadingNormal);
        tc0Direction = desc.asMeshLight.transform * tc0Direction;

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = normalize(tc0Direction - dotNT * shadingNormal);

        SurfacePoint &surfPt = result->surfPt;

        surfPt.position = position;
        surfPt.shadingFrame = ReferenceFrame(tc0Direction, shadingNormal);
        surfPt.isPoint = false;
        surfPt.atInfinity = false;
        surfPt.geometricNormal = geometricNormal;
        surfPt.u = b0;
        surfPt.v = b1;
        surfPt.texCoord = texCoord;
        surfPt.tc0Direction = tc0Direction;
    }
}
