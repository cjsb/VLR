#include "hip/hip_runtime.h"
﻿#include "kernel_common.cuh"

namespace VLR {
    // per GeometryInstance
    // closestHitProgramなどから呼ばれるdecodeHitPoint等で読み出すためにはGeometryInstanceレベルにバインドする必要がある。
    rtBuffer<Vertex> pv_vertexBuffer;
    rtBuffer<Triangle> pv_triangleBuffer;
    rtDeclareVariable(float, pv_sumImportances, , );

    rtDeclareVariable(optix::Ray, sm_ray, rtCurrentRay, );
    rtDeclareVariable(HitPointParameter, a_hitPointParam, attribute hitPointParam, );

    // Intersection Program
    RT_PROGRAM void intersectTriangle(int32_t primIdx) {
        const Triangle &triangle = pv_triangleBuffer[primIdx];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        // use a triangle intersection function defined in optix_math_namespace.h
        optix::float3 gn;
        float t;
        float b0, b1, b2;
        if (!intersect_triangle(sm_ray, asOptiXType(v0.position), asOptiXType(v1.position), asOptiXType(v2.position),
                                gn, t, b1, b2))
            return;

        if (!rtPotentialIntersection(t))
            return;

        b0 = 1.0f - b1 - b2;
        a_hitPointParam.b0 = b0;
        a_hitPointParam.b1 = b1;
        a_hitPointParam.primIndex = primIdx;

        const uint32_t materialIndex = 0;
        rtReportIntersection(materialIndex);
    }

    // Bounding Box Program
    RT_PROGRAM void calcBBoxForTriangle(int32_t primIdx, float result[6]) {
        const Triangle &triangle = pv_triangleBuffer[primIdx];
        const Point3D &p0 = pv_vertexBuffer[triangle.index0].position;
        const Point3D &p1 = pv_vertexBuffer[triangle.index1].position;
        const Point3D &p2 = pv_vertexBuffer[triangle.index2].position;

        //optix::Aabb* bbox = (optix::Aabb*)result;
        //*bbox = optix::Aabb(asOptiXType(p0), asOptiXType(p1), asOptiXType(p2));

        BoundingBox3D* bbox = (BoundingBox3D*)result;
        *bbox = BoundingBox3D(Point3D(INFINITY), Point3D(-INFINITY));
        bbox->unify(p0);
        bbox->unify(p1);
        bbox->unify(p2);
    }


    
    // bound
    RT_CALLABLE_PROGRAM void decodeHitPointForTriangle(const HitPointParameter &param, SurfacePoint* surfPt, float* areaPDF) {
        const Triangle &triangle = pv_triangleBuffer[param.primIndex];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        Normal3D geometricNormal = cross(v1.position - v0.position, v2.position - v0.position);
        float area = geometricNormal.length() / 2;
        geometricNormal /= 2 * area;

        float b0 = param.b0, b1 = param.b1, b2 = 1.0f - param.b0 - param.b1;
        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = normalize(b0 * v0.normal + b1 * v1.normal + b2 * v2.normal);
        Vector3D shadingTangent = normalize(b0 * v0.tangent + b1 * v1.tangent + b2 * v2.tangent);
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        //TexCoord2D dUV0m2 = v0.texCoord - v2.texCoord;
        //TexCoord2D dUV1m2 = v1.texCoord - v2.texCoord;
        //Vector3D dP0m2 = v0.position - v2.position;
        //Vector3D dP1m2 = v1.position - v2.position;

        //float invDetUV = 1.0f / (dUV0m2.u * dUV1m2.v - dUV0m2.v * dUV1m2.u);
        //Vector3D uDirection = invDetUV * Vector3D(dUV1m2.v * dP0m2.x - dUV0m2.v * dP1m2.x,
        //                                          dUV1m2.v * dP0m2.y - dUV0m2.v * dP1m2.y,
        //                                          dUV1m2.v * dP0m2.z - dUV0m2.v * dP1m2.z);
        //uDirection = normalize(cross(cross(shadingNormal, uDirection), shadingNormal));

        position = transform(RT_OBJECT_TO_WORLD, position);
        shadingNormal = normalize(transform(RT_OBJECT_TO_WORLD, shadingNormal));
        shadingTangent = normalize(transform(RT_OBJECT_TO_WORLD, shadingTangent));

        surfPt->position = position;
        surfPt->shadingFrame = ReferenceFrame(shadingTangent, shadingNormal);
        surfPt->atInfinity = false;

        surfPt->geometricNormal = normalize(transform(RT_OBJECT_TO_WORLD, geometricNormal));
        surfPt->u = b0;
        surfPt->v = b1;
        surfPt->texCoord = texCoord;
        //surfPt->tc0Direction = normalize(transform(RT_OBJECT_TO_WORLD, uDirection));

        float probLightPrim = area / pv_sumImportances;
        *areaPDF = probLightPrim / area;
    }

    RT_CALLABLE_PROGRAM TexCoord2D decodeTexCoordForTriangle(const HitPointParameter &param) {
        const Triangle &triangle = pv_triangleBuffer[param.primIndex];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        float b0 = param.b0, b1 = param.b1, b2 = 1.0f - param.b0 - param.b1;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        return texCoord;
    }



    RT_CALLABLE_PROGRAM void sampleTriangleMesh(const SurfaceLightDescriptor::Body &desc, const SurfaceLightPosSample &sample, SurfaceLightPosQueryResult* result) {
        float primProb;
        uint32_t primIdx = desc.primDistribution.sample(sample.uElem, &primProb);

        result->materialIndex = desc.materialIndex;

        const Triangle &triangle = desc.triangleBuffer[primIdx];
        const Vertex &v0 = desc.vertexBuffer[triangle.index0];
        const Vertex &v1 = desc.vertexBuffer[triangle.index1];
        const Vertex &v2 = desc.vertexBuffer[triangle.index2];

        Normal3D geometricNormal = cross(v1.position - v0.position, v2.position - v0.position);
        float area = geometricNormal.length() / 2;
        geometricNormal /= 2 * area;

        float b0, b1, b2;
        uniformSampleTriangle(sample.uPos[0], sample.uPos[1], &b0, &b1);
        b2 = 1.0f - b0 - b1;

        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D shadingTangent = b0 * v0.tangent + b1 * v1.tangent + b2 * v2.tangent;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        ReferenceFrame shadingFrame;
        shadingFrame.z = normalize(desc.transform * shadingNormal);
        shadingFrame.x = normalize(desc.transform * shadingTangent);
        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingFrame.z, shadingFrame.x);
        if (std::fabs(dotNT) >= 0.01f)
            shadingFrame.x = normalize(shadingFrame.x - dotNT * shadingFrame.z);
        shadingFrame.y = cross(shadingFrame.z, shadingFrame.x);

        SurfacePoint &surfPt = result->surfPt;

        surfPt.position = desc.transform * position;
        surfPt.shadingFrame = shadingFrame;
        surfPt.isPoint = false;
        surfPt.atInfinity = false;

        surfPt.geometricNormal = normalize(desc.transform * geometricNormal);
        surfPt.u = b0;
        surfPt.v = b1;
        surfPt.texCoord = texCoord;

        result->areaPDF = primProb / area;
        result->posType = DirectionType::Emission() | DirectionType::LowFreq();



        //result->surfPt.isPoint = false;
        //result->surfPt.atInfinity = false;
        //result->surfPt.geometricNormal = Normal3D(0, -1, 0);
        //result->surfPt.shadingFrame = ReferenceFrame(Vector3D(1, 0, 0), Vector3D(0, 0, 1), Normal3D(0, -1, 0));
        //result->surfPt.u = sample.uPos[0];
        //result->surfPt.v = sample.uPos[1];
        //result->surfPt.position = Point3D(-0.5f, 2.899f, -0.5f) + result->surfPt.u * Vector3D(1, 0, 0) + result->surfPt.v * Vector3D(0, 0, 1);
        //result->surfPt.texCoord = TexCoord2D(result->surfPt.u, result->surfPt.v);
        //result->areaPDF = 1.0f / 1.0f;
        //result->posType = DirectionType::Emission() | DirectionType::LowFreq();
        //result->materialIndex = 5;
    }
}
