#include "hip/hip_runtime.h"
#include "kernel_common.cuh"

namespace VLR {
    RT_CALLABLE_PROGRAM float FloatShaderNode_float(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const FloatShaderNode &nodeData = *(const FloatShaderNode*)rawNodeData;
        return calcNode<float>(nodeData.node0, nodeData.imm0, surfPt);
    }



    RT_CALLABLE_PROGRAM float Float2ShaderNode_float(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float2ShaderNode &nodeData = *(const Float2ShaderNode*)rawNodeData;
        if (option == 0)
            return calcNode<float>(nodeData.node0, nodeData.imm0, surfPt);
        else if (option == 1)
            return calcNode<float>(nodeData.node1, nodeData.imm1, surfPt);
        return 0.0f;
    }
    
    RT_CALLABLE_PROGRAM optix::float2 Float2ShaderNode_float2(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float2ShaderNode &nodeData = *(const Float2ShaderNode*)rawNodeData;
        return optix::make_float2(calcNode<float>(nodeData.node0, nodeData.imm0, surfPt),
                                  calcNode<float>(nodeData.node1, nodeData.imm1, surfPt));
    }



    RT_CALLABLE_PROGRAM float Float3ShaderNode_float(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float3ShaderNode &nodeData = *(const Float3ShaderNode*)rawNodeData;
        if (option == 0)
            return calcNode<float>(nodeData.node0, nodeData.imm0, surfPt);
        else if (option == 1)
            return calcNode<float>(nodeData.node1, nodeData.imm1, surfPt);
        else if (option == 2)
            return calcNode<float>(nodeData.node2, nodeData.imm2, surfPt);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float3ShaderNode_float2(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float3ShaderNode &nodeData = *(const Float3ShaderNode*)rawNodeData;
        if (option == 0)
            return optix::make_float2(calcNode<float>(nodeData.node0, nodeData.imm0, surfPt),
                                      calcNode<float>(nodeData.node1, nodeData.imm1, surfPt));
        else if (option == 1)
            return optix::make_float2(calcNode<float>(nodeData.node1, nodeData.imm1, surfPt),
                                      calcNode<float>(nodeData.node2, nodeData.imm2, surfPt));
        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Float3ShaderNode_float3(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float3ShaderNode &nodeData = *(const Float3ShaderNode*)rawNodeData;
        return optix::make_float3(calcNode<float>(nodeData.node0, nodeData.imm0, surfPt),
                                  calcNode<float>(nodeData.node1, nodeData.imm1, surfPt),
                                  calcNode<float>(nodeData.node2, nodeData.imm2, surfPt));
    }



    RT_CALLABLE_PROGRAM float Float4ShaderNode_float(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float4ShaderNode &nodeData = *(const Float4ShaderNode*)rawNodeData;
        if (option == 0)
            return calcNode<float>(nodeData.node0, nodeData.imm0, surfPt);
        else if (option == 1)
            return calcNode<float>(nodeData.node1, nodeData.imm1, surfPt);
        else if (option == 2)
            return calcNode<float>(nodeData.node2, nodeData.imm2, surfPt);
        else if (option == 3)
            return calcNode<float>(nodeData.node3, nodeData.imm3, surfPt);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float4ShaderNode_float2(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float4ShaderNode &nodeData = *(const Float4ShaderNode*)rawNodeData;
        if (option == 0)
            return optix::make_float2(calcNode<float>(nodeData.node0, nodeData.imm0, surfPt),
                                      calcNode<float>(nodeData.node1, nodeData.imm1, surfPt));
        else if (option == 1)
            return optix::make_float2(calcNode<float>(nodeData.node1, nodeData.imm1, surfPt),
                                      calcNode<float>(nodeData.node2, nodeData.imm2, surfPt));
        else if (option == 2)
            return optix::make_float2(calcNode<float>(nodeData.node2, nodeData.imm2, surfPt),
                                      calcNode<float>(nodeData.node3, nodeData.imm3, surfPt));
        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Float4ShaderNode_float3(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float4ShaderNode &nodeData = *(const Float4ShaderNode*)rawNodeData;
        if (option == 0)
            return optix::make_float3(calcNode<float>(nodeData.node0, nodeData.imm0, surfPt),
                                      calcNode<float>(nodeData.node1, nodeData.imm1, surfPt),
                                      calcNode<float>(nodeData.node2, nodeData.imm2, surfPt));
        else if (option == 1)
            return optix::make_float3(calcNode<float>(nodeData.node1, nodeData.imm1, surfPt),
                                      calcNode<float>(nodeData.node2, nodeData.imm2, surfPt),
                                      calcNode<float>(nodeData.node3, nodeData.imm3, surfPt));
        return optix::make_float3(0.0f, 0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float4 Float4ShaderNode_float4(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Float4ShaderNode &nodeData = *(const Float4ShaderNode*)rawNodeData;
        return optix::make_float4(calcNode<float>(nodeData.node0, nodeData.imm0, surfPt),
                                  calcNode<float>(nodeData.node1, nodeData.imm1, surfPt),
                                  calcNode<float>(nodeData.node2, nodeData.imm2, surfPt),
                                  calcNode<float>(nodeData.node3, nodeData.imm3, surfPt));
    }



    RT_CALLABLE_PROGRAM Point3D OffsetAndScaleUVTextureMap2DShaderNode_TexCoord(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const OffsetAndScaleUVTextureMap2DShaderNode &nodeData = *(const OffsetAndScaleUVTextureMap2DShaderNode*)rawNodeData;
        return Point3D(nodeData.scale[0] * surfPt.texCoord.u + nodeData.offset[0],
                       nodeData.scale[1] * surfPt.texCoord.v + nodeData.offset[1],
                       0.0f);
    }



    RT_CALLABLE_PROGRAM RGBSpectrum ConstantTextureShaderNode_RGBSpectrum(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const ConstantTextureShaderNode &nodeData = *(const ConstantTextureShaderNode*)rawNodeData;
        return nodeData.spectrum;
    }

    RT_CALLABLE_PROGRAM float ConstantTextureShaderNode_Alpha(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const ConstantTextureShaderNode &nodeData = *(const ConstantTextureShaderNode*)rawNodeData;
        return nodeData.alpha;
    }



    RT_CALLABLE_PROGRAM RGBSpectrum Image2DTextureShaderNode_RGBSpectrum(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Image2DTextureShaderNode &nodeData = *(const Image2DTextureShaderNode*)rawNodeData;

        Point3D texCoord = calcNode<Point3D>(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt);
        optix::float4 texValue = optix::rtTex2D<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y);

        return RGBSpectrum(texValue.x, texValue.y, texValue.z);
    }

    RT_CALLABLE_PROGRAM float Image2DTextureShaderNode_float(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Image2DTextureShaderNode &nodeData = *(const Image2DTextureShaderNode*)rawNodeData;

        Point3D texCoord = calcNode<Point3D>(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt);
        optix::float4 texValue = optix::rtTex2D<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y);

        if (option == 0)
            return texValue.x;
        else if (option == 1)
            return texValue.y;
        else if (option == 2)
            return texValue.z;
        else if (option == 3)
            return texValue.w;

        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Image2DTextureShaderNode_float2(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Image2DTextureShaderNode &nodeData = *(const Image2DTextureShaderNode*)rawNodeData;

        Point3D texCoord = calcNode<Point3D>(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt);
        optix::float4 texValue = optix::rtTex2D<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y);

        if (option == 0)
            return optix::make_float2(texValue.x, texValue.y);
        else if (option == 1)
            return optix::make_float2(texValue.y, texValue.z);
        else if (option == 2)
            return optix::make_float2(texValue.z, texValue.w);

        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Image2DTextureShaderNode_float3(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Image2DTextureShaderNode &nodeData = *(const Image2DTextureShaderNode*)rawNodeData;

        Point3D texCoord = calcNode<Point3D>(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt);
        optix::float4 texValue = optix::rtTex2D<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y);

        if (option == 0)
            return optix::make_float3(texValue.x, texValue.y, texValue.z);
        else if (option == 1)
            return optix::make_float3(texValue.y, texValue.z, texValue.w);

        return optix::make_float3(0.0f, 0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float4 Image2DTextureShaderNode_float4(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const Image2DTextureShaderNode &nodeData = *(const Image2DTextureShaderNode*)rawNodeData;

        Point3D texCoord = calcNode<Point3D>(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt);
        optix::float4 texValue = optix::rtTex2D<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y);

        return texValue;
    }



    RT_CALLABLE_PROGRAM RGBSpectrum EnvironmentTextureShaderNode_RGBSpectrum(const uint32_t* rawNodeData, uint32_t option, const SurfacePoint &surfPt) {
        const EnvironmentTextureShaderNode &nodeData = *(const EnvironmentTextureShaderNode*)rawNodeData;

        Point3D texCoord = calcNode<Point3D>(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt);
        optix::float4 texValue = optix::rtTex2D<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y);

        return RGBSpectrum(texValue.x, texValue.y, texValue.z);
    }
}
